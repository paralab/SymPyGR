#include "hip/hip_runtime.h"
#include "bssneqn_solve.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

__constant__ double ETA_CONST=0.1;
__constant__ double ETA_R0=0.1;
__constant__ double ETA_DAMPING_EXP=0.1;
__constant__ unsigned int lambda[4]={1,2,3,4};
__constant__ double lambda_f[2]={0.8,0.9};

__global__ void cuda_bssn_eqns_points(int * dev_offset, int * dev_sz, double * dev_pmin, double * dev_dy_hz, double * dev_dy_hy, double * dev_dy_hx, 
    double * dev_var_in, double * dev_var_out,
    #include "list_of_para.h"
    )
{
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

    double z = dev_pmin[2] + *dev_dy_hz*k;
    double y = dev_pmin[1] + *dev_dy_hy*j;
    double x = dev_pmin[0] + *dev_dy_hx*i;

    int pp = i + nx*(j + ny*k);
    double r_coord = sqrt(x*x + y*y + z*z);
    double eta = ETA_CONST;
    if (r_coord >= ETA_R0) {
        eta *= pow( (ETA_R0/r_coord), ETA_DAMPING_EXP);
    }
    #include "cuda_bssneqs.h"
}

void calc_bssn_eqns(const unsigned int * sz, int * dev_sz, double * dev_pmin, double * dev_dy_hz, double * dev_dy_hy, double * dev_dy_hx, 
double * dev_var_in, double * dev_var_out, 
#include "list_of_para.h"
)
{
    const int ie = sz[0] - 3;//x direction
    const int je = sz[1] - 3;//y direction
    const int ke = sz[2] - 3;//z direction
  
    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
     
    int requiredBlocks = (5+maximumIterations) / 6;

    cuda_bssn_eqns_points <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                      dim3((ie + requiredBlocks -1)/requiredBlocks,
                      (je + requiredBlocks -1)/requiredBlocks, 
                      (ke + requiredBlocks -1)/requiredBlocks) >>> (0, dev_sz, dev_pmin, dev_dy_hz, dev_dy_hy, dev_dy_hx, dev_var_in, dev_var_out,
                        #include "list_of_args.h"
                    );
    CHECK_ERROR(hipGetLastError(), "cuda_bssn_eqns_points Kernel launch failed");
    
}
