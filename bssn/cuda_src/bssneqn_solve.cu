#include "hip/hip_runtime.h"
#include "bssneqn_solve.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

int threads_per_block_cpu=250;
int blocks_cpu=50;

__constant__ int threads_per_block=250;
__constant__ int blocks=50;

__constant__ double ETA_CONST=0.1;
__constant__ double ETA_R0=0.1;
__constant__ double ETA_DAMPING_EXP=0.1;
__constant__ unsigned int lambda[4]={1,2,3,4};
__constant__ double lambda_f[2]={0.8,0.9};

__global__ void cuda_bssn_eqns_points(int * dev_offset, int * dev_sz, double * dev_pmin, double * dev_dy_hz, double * dev_dy_hy, double * dev_dy_hx, 
    double * dev_var_in, double * dev_var_out,
    #include "list_of_para.h"
    )
{
    int id = *dev_offset + blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(dev_sz[0]-6) + 3;
    int j = ((id/(dev_sz[0]-6))%(dev_sz[1]-6)) + 3;
    int k = (id/(dev_sz[2]-6)/(dev_sz[1]-6)) + 3;

    if (k>=dev_sz[2]-3) return;

    double z = dev_pmin[2] + *dev_dy_hz*k;
    double y = dev_pmin[1] + *dev_dy_hy*j;
    double x = dev_pmin[0] + *dev_dy_hx*i;

    int pp = i + (dev_sz[0])*(j + (dev_sz[1])*k);
    double r_coord = sqrt(x*x + y*y + z*z);
    double eta = ETA_CONST;
    if (r_coord >= ETA_R0) {
        eta *= pow( (ETA_R0/r_coord), ETA_DAMPING_EXP);
    }
    #include "cuda_bssneqs.h"
}

void calc_bssn_eqns(const unsigned int * sz, int * dev_sz, double * dev_pmin, double * dev_dy_hz, double * dev_dy_hy, double * dev_dy_hx, 
double * dev_var_in, double * dev_var_out, 
#include "list_of_para.h"
)
{
    int total_points = (sz[2]-6)*(sz[1]-6)*(sz[0]-6);

    int points_at_once = threads_per_block_cpu*blocks_cpu;
    int loops = ceil(1.0*total_points/points_at_once);

    int * dev_offset;
    CHECK_ERROR(hipMalloc((void **) &dev_offset, sizeof(int)), "dev_offset hipMalloc in bssneqn_solve.cu");
    for(int i=0; i<loops; i++){
        int offset = i*points_at_once;
        CHECK_ERROR(hipMemcpy(dev_offset, &offset, sizeof(int), hipMemcpyHostToDevice), "dev_offset hipMemcpy in bssneqn_solve.cu");

        cuda_bssn_eqns_points<<< blocks_cpu, threads_per_block_cpu >>>(dev_offset, dev_sz, dev_pmin, dev_dy_hz, dev_dy_hy, dev_dy_hx, dev_var_in, dev_var_out,
            #include "list_of_args.h"
        );
        CHECK_ERROR(hipGetLastError(), "cuda_bssn_eqns_points Kernel launch failed");
    } 
    // CHECK_ERROR(hipFree(dev_offset), "dev_offset hipFree in bssneqn_solve.cu");
}
