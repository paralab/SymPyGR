#include "hip/hip_runtime.h"
#include "bssneqn_solve.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

int threads_per_block_cpu=250;
int blocks_cpu=50;

__constant__ int threads_per_block=250;
__constant__ int blocks=50;

__constant__ double ETA_CONST=0.1;
__constant__ double ETA_R0=0.1;
__constant__ double ETA_DAMPING_EXP=0.1;
__constant__ unsigned int lambda[4]={1,2,3,4};
__constant__ double lambda_f[2]={0.8,0.9};

__global__ void cuda_bssn_eqns_points(int * dev_offset, int * dev_sz, double * dev_pmin, double * dev_dy_hz, double * dev_dy_hy, double * dev_dy_hx, 
    double * dev_var_in, double * dev_var_out,
    #include "list_of_para.h"
    )
{
    int id = *dev_offset + blockIdx.x*threads_per_block + threadIdx.x;

    int i = id%(dev_sz[0]-6) + 3;
    int j = ((id/(dev_sz[0]-6))%(dev_sz[1]-6)) + 3;
    int k = (id/(dev_sz[2]-6)/(dev_sz[1]-6)) + 3;

    if (k>=dev_sz[2]-3) return;

    double z = dev_pmin[2] + *dev_dy_hz*k;
    double y = dev_pmin[1] + *dev_dy_hy*j;
    double x = dev_pmin[0] + *dev_dy_hx*i;

    int pp = i + (dev_sz[0])*(j + (dev_sz[1])*k);
    double r_coord = sqrt(x*x + y*y + z*z);
    double eta = ETA_CONST;
    if (r_coord >= ETA_R0) {
        eta *= pow( (ETA_R0/r_coord), ETA_DAMPING_EXP);
    }
    #include "cuda_bssneqs.h"
}

void calc_bssn_eqns(const unsigned int * sz, int * dev_sz, double * dev_pmin, double * dev_dy_hz, double * dev_dy_hy, double * dev_dy_hx, 
double * dev_var_in, double * dev_var_out, 
#include "list_of_para.h"
)
{
    int total_points = (sz[2]-6)*(sz[1]-6)*(sz[0]-6);

    int points_at_once = threads_per_block_cpu*blocks_cpu;
    int loops = ceil(1.0*total_points/points_at_once);

    hipError_t cudaStatus;
    // full
    for(int i=0; i<loops; i++){
        int offset = i*points_at_once;

        int * dev_offset;
        cudaStatus = hipMalloc((void **) &dev_offset, sizeof(int));
        if (cudaStatus != hipSuccess) {fprintf(stderr, "dev_offset hipMalloc failed!\n"); return;}
        cudaStatus = hipMemcpy(dev_offset, &offset, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {fprintf(stderr, "dev_offset hipMemcpy failed!\n"); return;}

        cuda_bssn_eqns_points<<< blocks_cpu, threads_per_block_cpu >>>(dev_offset, dev_sz, dev_pmin, dev_dy_hz, dev_dy_hy, dev_dy_hx, dev_var_in, dev_var_out,
            #include "list_of_args.h"
        );
        
        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_bssn_eqns_points Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        hipFree(dev_offset);
    } 
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_bssn_eqns_points kernals!\n", cudaStatus);
        return;
    }
}
