#include "hip/hip_runtime.h"
#include "rhs_cuda.h"
#include "bssneqn_solve.h"

#include <iostream>
#include <stdio.h>

enum VAR_CU {U_ALPHA=0,U_CHI,U_K,U_GT0,U_GT1,U_GT2,U_BETA0,U_BETA1,U_BETA2,U_B0,U_B1,U_B2,U_SYMGT0,U_SYMGT1,U_SYMGT2,U_SYMGT3,U_SYMGT4,U_SYMGT5,U_SYMAT0,U_SYMAT1,U_SYMAT2,U_SYMAT3,U_SYMAT4,U_SYMAT5};

double ETA_CONST_CUDA=0.1;
double ETA_R0_CUDA=0.1;
double ETA_DAMPING_EXP_CUDA=0.1;
double KO_DISS_SIGMA_CUDA=1e-4;
unsigned int BSSN_LAMBDA_CUDA[4]={1,2,3,4};
double BSSN_LAMBDA_F_CUDA[2]={0.8,0.9};

__global__ void example_kernal(double * val){
    // Eminda you can use this if it is required
    //test GPU mem values
}

void cuda_bssnrhs(double * dev_var_out, double * dev_var_in, const unsigned int unzip_dof, 
const unsigned int& offset, const double *pmin, const double *pmax, const unsigned int *sz, 
const unsigned int& bflag)
{
    hipError_t cudaStatus;

    int alphaInt = (VAR_CU::U_ALPHA) * unzip_dof + offset;
    int chiInt = (VAR_CU::U_CHI) * unzip_dof + offset;
    int KInt = (VAR_CU::U_K) * unzip_dof + offset;
    int gt0Int = (VAR_CU::U_SYMGT0) * unzip_dof + offset;
    int gt1Int = (VAR_CU::U_SYMGT1) * unzip_dof + offset;
    int gt2Int =  (VAR_CU::U_SYMGT2) * unzip_dof + offset;
    int gt3Int =(VAR_CU::U_SYMGT3) * unzip_dof + offset;
    int gt4Int = (VAR_CU::U_SYMGT4) * unzip_dof + offset;
    int gt5Int = (VAR_CU::U_SYMGT5) * unzip_dof + offset;
    int beta0Int = (VAR_CU::U_BETA0) * unzip_dof + offset;
    int beta1Int = (VAR_CU::U_BETA1) * unzip_dof + offset;
    int beta2Int =(VAR_CU::U_BETA2) * unzip_dof + offset;
    int At0Int = (VAR_CU::U_SYMAT0) * unzip_dof + offset;
    int At1Int = (VAR_CU::U_SYMAT1) * unzip_dof + offset;
    int At2Int = (VAR_CU::U_SYMAT2) * unzip_dof + offset;
    int At3Int = (VAR_CU::U_SYMAT3) * unzip_dof + offset;
    int At4Int = (VAR_CU::U_SYMAT4) * unzip_dof + offset;
    int At5Int = (VAR_CU::U_SYMAT5) * unzip_dof + offset;
    int Gt0Int = (VAR_CU::U_GT0) * unzip_dof + offset;
    int Gt1Int = (VAR_CU::U_GT1) * unzip_dof + offset;
    int Gt2Int = (VAR_CU::U_GT2) * unzip_dof + offset;
    int B0Int = (VAR_CU::U_B0) * unzip_dof + offset;
    int B1Int = (VAR_CU::U_B1) * unzip_dof + offset;
    int B2Int = (VAR_CU::U_B2) * unzip_dof + offset;
    
    int lbflag = (bflag & (1u<<OCT_DIR_LEFT)) ? 1 : 0;
    int rbflag = (bflag & (1u<<OCT_DIR_RIGHT)) ? 1 : 0;

    double hx = (pmax[0] - pmin[0]) / (sz[0] - 1);
    double hy = (pmax[1] - pmin[1]) / (sz[1] - 1);
    double hz = (pmax[2] - pmin[2]) / (sz[2] - 1);

    // Send above values to GPU memory
    #include "bssnrhs_cuda_offset_malloc.h"

    double * dev_dy_hx; //similar to hx in cpu code
    cudaStatus = hipMalloc((void **) &dev_dy_hx, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hx, &hx, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMemcpy failed!\n"); return;}

    double * dev_dy_hy;
    cudaStatus = hipMalloc((void **) &dev_dy_hy, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hy, &hy, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMemcpy failed!\n"); return;}

    double * dev_dy_hz;
    cudaStatus = hipMalloc((void **) &dev_dy_hz, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hz, &hz, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMemcpy failed!\n"); return;}

    int * dev_sz;
    cudaStatus = hipMalloc((void **) &dev_sz, 3*sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_sz, sz, 3*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMemcpy failed!\n"); return;}

    int * dev_zero;
    cudaStatus = hipMalloc((void **) &dev_zero, sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "0 hipMalloc failed!\n"); return;}

    double *dev_pmin;
    cudaStatus = hipMalloc((void **) &dev_pmin, sizeof(pmin)*sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmin hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_pmin, pmin, sizeof(pmin)*sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmin hipMemcpy failed!\n"); return;}

    const unsigned int lambda[4] = {BSSN_LAMBDA_CUDA[0], BSSN_LAMBDA_CUDA[1],
        BSSN_LAMBDA_CUDA[2], BSSN_LAMBDA_CUDA[3]};
    const double lambda_f[2] = {BSSN_LAMBDA_F_CUDA[0], BSSN_LAMBDA_F_CUDA[1]};

    unsigned int *dev_lambda;
    cudaStatus = hipMalloc((void **) &dev_lambda, sizeof(lambda)*sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "lambda hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_lambda, lambda, sizeof(lambda)*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "lambda hipMemcpy failed!\n"); return;}

    double *dev_lambda_f;
    cudaStatus = hipMalloc((void **) &dev_lambda_f, sizeof(lambda_f)*sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "lambda_f hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_lambda_f, lambda_f, sizeof(lambda_f)*sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "lambda_f hipMemcpy failed!\n"); return;}

    // Allocate memory to store the output of derivs
    unsigned int n = sz[0]*sz[1]*sz[2];
    int size = n * sizeof(double);

    #include "bssnrhs_cuda_malloc.h"
    #include "bssnrhs_cuda_malloc_adv.h"

    bssn::timer::t_deriv.start();

    // Deriv calls are follows
    #include "bssnrhs_cuda_derivs.h"
    #include "bssnrhs_cuda_derivs_adv.h"

    bssn::timer::t_deriv.stop();

//     int sizeArray=(sz[2]-3)*(sz[1]-3)*(sz[0]-3);
//     callculateBSSN_EQ(
//   grad_0_alpha,
//   grad_1_alpha,
//   grad_2_alpha,
//   grad_0_beta0,
//   grad_1_beta0,
//   grad_2_beta0,
//   grad_0_beta1,
//   grad_1_beta1,
//   grad_2_beta1,
//   grad_0_beta2,
//   grad_1_beta2,
//   grad_2_beta2,
//   grad_0_B0,
//   grad_1_B0,
//   grad_2_B0,
//   grad_0_B1,
//   grad_1_B1,
//   grad_2_B1,
//   grad_0_B2,
//   grad_1_B2,
//   grad_2_B2,
//   grad_0_chi,
//   grad_1_chi,
//   grad_2_chi,
//   grad_0_Gt0,
//   grad_1_Gt0,
//   grad_2_Gt0,
//   grad_0_Gt1,
//   grad_1_Gt1,
//   grad_2_Gt1,
//   grad_0_Gt2,
//   grad_1_Gt2,
//   grad_2_Gt2,
//   grad_0_K,
//   grad_1_K,
//   grad_2_K,
//   grad_0_gt0,
//   grad_1_gt0,
//   grad_2_gt0,
//   grad_0_gt1,
//   grad_1_gt1,
//   grad_2_gt1,
//   grad_0_gt2,
//   grad_1_gt2,
//   grad_2_gt2,
//   grad_0_gt3,
//   grad_1_gt3,
//   grad_2_gt3,
//   grad_0_gt4,
//   grad_1_gt4,
//   grad_2_gt4,
//   grad_0_gt5,
//   grad_1_gt5,
//   grad_2_gt5,
//   grad_0_At0,
//   grad_1_At0,
//   grad_2_At0,
//   grad_0_At1,
//   grad_1_At1,
//   grad_2_At1,
//   grad_0_At2,
//   grad_1_At2,
//   grad_2_At2,
//   grad_0_At3,
//   grad_1_At3,
//   grad_2_At3,
//   grad_0_At4,
//   grad_1_At4,
//   grad_2_At4,
//   grad_0_At5,
//   grad_1_At5,
//   grad_2_At5,
//   grad2_0_0_gt0,
//   grad2_0_1_gt0,
//   grad2_0_2_gt0,
//   grad2_1_1_gt0,
//   grad2_1_2_gt0,
//   grad2_2_2_gt0,
//   grad2_0_0_gt1,
//   grad2_0_1_gt1,
//   grad2_0_2_gt1,
//   grad2_1_1_gt1,
//   grad2_1_2_gt1,
//   grad2_2_2_gt1,
//   grad2_0_0_gt2,
//   grad2_0_1_gt2,
//   grad2_0_2_gt2,
//   grad2_1_1_gt2,
//   grad2_1_2_gt2,
//   grad2_2_2_gt2,
//   grad2_0_0_gt3,
//   grad2_0_1_gt3,
//   grad2_0_2_gt3,
//   grad2_1_1_gt3,
//   grad2_1_2_gt3,
//   grad2_2_2_gt3,
//   grad2_0_0_gt4,
//   grad2_0_1_gt4,
//   grad2_0_2_gt4,
//   grad2_1_1_gt4,
//   grad2_1_2_gt4,
//   grad2_2_2_gt4,
//   grad2_0_0_gt5,
//   grad2_0_1_gt5,
//   grad2_0_2_gt5,
//   grad2_1_1_gt5,
//   grad2_1_2_gt5,
//   grad2_2_2_gt5,
//   grad2_0_0_chi,
//   grad2_0_1_chi,
//   grad2_0_2_chi,
//   grad2_1_1_chi,
//   grad2_1_2_chi,
//   grad2_2_2_chi,
//   grad2_0_0_alpha,
//   grad2_0_1_alpha,
//   grad2_0_2_alpha,
//   grad2_1_1_alpha,
//   grad2_1_2_alpha,
//   grad2_2_2_alpha,
//   grad2_0_0_beta0,
//   grad2_0_1_beta0,
//   grad2_0_2_beta0,
//   grad2_1_1_beta0,
//   grad2_1_2_beta0,
//   grad2_2_2_beta0,
//   grad2_0_0_beta1,
//   grad2_0_1_beta1,
//   grad2_0_2_beta1,
//   grad2_1_1_beta1,
//   grad2_1_2_beta1,
//   grad2_2_2_beta1,
//   grad2_0_0_beta2,
//   grad2_0_1_beta2,
//   grad2_0_2_beta2,
//   grad2_1_1_beta2,
//   grad2_1_2_beta2,
//   grad2_2_2_beta2,
//   agrad_0_gt0,
//   agrad_1_gt0,
//   agrad_2_gt0,
//   agrad_0_gt1,
//   agrad_1_gt1,
//   agrad_2_gt1,
//   agrad_0_gt2,
//   agrad_1_gt2,
//   agrad_2_gt2,
//   agrad_0_gt3,
//   agrad_1_gt3,
//   agrad_2_gt3,
//   agrad_0_gt4,
//   agrad_1_gt4,
//   agrad_2_gt4,
//   agrad_0_gt5,
//   agrad_1_gt5,
//   agrad_2_gt5,
//   agrad_0_At0,
//   agrad_1_At0,
//   agrad_2_At0,
//   agrad_0_At1,
//   agrad_1_At1,
//   agrad_2_At1,
//   agrad_0_At2,
//   agrad_1_At2,
//   agrad_2_At2,
//   agrad_0_At3,
//   agrad_1_At3,
//   agrad_2_At3,
//   agrad_0_At4,
//   agrad_1_At4,
//   agrad_2_At4,
//   agrad_0_At5,
//   agrad_1_At5,
//   agrad_2_At5,
//   agrad_0_alpha,
//   agrad_1_alpha,
//   agrad_2_alpha,
//   agrad_0_beta0,
//   agrad_1_beta0,
//   agrad_2_beta0,
//   agrad_0_beta1,
//   agrad_1_beta1,
//   agrad_2_beta1,
//   agrad_0_beta2,
//   agrad_1_beta2,
//   agrad_2_beta2,
//   agrad_0_chi,
//   agrad_1_chi,
//   agrad_2_chi,
//   agrad_0_Gt0,
//   agrad_1_Gt0,
//   agrad_2_Gt0,
//   agrad_0_Gt1,
//   agrad_1_Gt1,
//   agrad_2_Gt1,
//   agrad_0_Gt2,
//   agrad_1_Gt2,
//   agrad_2_Gt2,
//   agrad_0_K,
//   agrad_1_K,
//   agrad_2_K,
//   agrad_0_B0,
//   agrad_1_B0,
//   agrad_2_B0,
//   agrad_0_B1,
//   agrad_1_B1,
//   agrad_2_B1,
//   agrad_0_B2,
//   agrad_1_B2,
//   agrad_2_B2,
//   dev_alphaInt,
//   dev_chiInt,
//   dev_KInt,
//   dev_gt0Int,
//   dev_gt1Int,
//   dev_gt2Int,
//   dev_gt3Int,
//   dev_gt4Int,
//   dev_gt5Int,
//   dev_beta0Int,
//   dev_beta1Int,
//   dev_beta2Int,
//   dev_At0Int,
//   dev_At1Int,
//   dev_At2Int,
//   dev_At3Int,
//   dev_At4Int,
//   dev_At5Int,
//   dev_Gt0Int,
//   dev_Gt1Int,
//   dev_Gt2Int,
//   dev_B0Int,
//   dev_B1Int,
//   dev_B2Int,
//   dev_lambda,
//   dev_lambda_f,
//   pmin,
//   dev_sz,
//   dev_dy_hx,
//   dev_dy_hy,
//   dev_dy_hz,
//   dev_var_in,
//   dev_var_out,
//   &sizeArray
// );
    // Free up GPU memory
    #include "bssnrhs_cuda_offset_demalloc.h"
    #include "bssnrhs_cuda_mdealloc.h"
    #include "bssnrhs_cuda_mdealloc_adv.h"
    hipFree(dev_dy_hx);
    hipFree(dev_dy_hy);
    hipFree(dev_dy_hz);
    hipFree(dev_sz);
    hipFree(dev_zero);
    hipFree(dev_pmin);
    hipFree(dev_lambda);
    hipFree(dev_lambda_f);
}