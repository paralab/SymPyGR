#include "hip/hip_runtime.h"
#include "rhs_cuda.h"
#include "bssneqn_solve.h"

#include <iostream>
#include <stdio.h>


enum VAR_CU {U_ALPHA=0,U_CHI,U_K,U_GT0,U_GT1,U_GT2,U_BETA0,U_BETA1,U_BETA2,U_B0,U_B1,U_B2,U_SYMGT0,U_SYMGT1,U_SYMGT2,U_SYMGT3,U_SYMGT4,U_SYMGT5,U_SYMAT0,U_SYMAT1,U_SYMAT2,U_SYMAT3,U_SYMAT4,U_SYMAT5};

void cuda_bssnrhs(double * dev_var_out, double * dev_var_in, const unsigned int unzip_dof, 
const unsigned int& offset, const double *pmin, const double *pmax, const unsigned int *sz, 
const unsigned int& bflag)
{ 
    int alphaInt = (VAR_CU::U_ALPHA) * unzip_dof + offset;
    int chiInt = (VAR_CU::U_CHI) * unzip_dof + offset;
    int KInt = (VAR_CU::U_K) * unzip_dof + offset;
    int gt0Int = (VAR_CU::U_SYMGT0) * unzip_dof + offset;
    int gt1Int = (VAR_CU::U_SYMGT1) * unzip_dof + offset;
    int gt2Int =  (VAR_CU::U_SYMGT2) * unzip_dof + offset;
    int gt3Int =(VAR_CU::U_SYMGT3) * unzip_dof + offset;
    int gt4Int = (VAR_CU::U_SYMGT4) * unzip_dof + offset;
    int gt5Int = (VAR_CU::U_SYMGT5) * unzip_dof + offset;
    int beta0Int = (VAR_CU::U_BETA0) * unzip_dof + offset;
    int beta1Int = (VAR_CU::U_BETA1) * unzip_dof + offset;
    int beta2Int =(VAR_CU::U_BETA2) * unzip_dof + offset;
    int At0Int = (VAR_CU::U_SYMAT0) * unzip_dof + offset;
    int At1Int = (VAR_CU::U_SYMAT1) * unzip_dof + offset;
    int At2Int = (VAR_CU::U_SYMAT2) * unzip_dof + offset;
    int At3Int = (VAR_CU::U_SYMAT3) * unzip_dof + offset;
    int At4Int = (VAR_CU::U_SYMAT4) * unzip_dof + offset;
    int At5Int = (VAR_CU::U_SYMAT5) * unzip_dof + offset;
    int Gt0Int = (VAR_CU::U_GT0) * unzip_dof + offset;
    int Gt1Int = (VAR_CU::U_GT1) * unzip_dof + offset;
    int Gt2Int = (VAR_CU::U_GT2) * unzip_dof + offset;
    int B0Int = (VAR_CU::U_B0) * unzip_dof + offset;
    int B1Int = (VAR_CU::U_B1) * unzip_dof + offset;
    int B2Int = (VAR_CU::U_B2) * unzip_dof + offset;

    double hx = (pmax[0] - pmin[0]) / (sz[0] - 1);
    double hy = (pmax[1] - pmin[1]) / (sz[1] - 1);
    double hz = (pmax[2] - pmin[2]) / (sz[2] - 1);

    bssn::timer::t_mem_handling_gpu.start();
    // Send above values to GPU memory
    hipError_t cudaStatus;
    #include "bssnrhs_cuda_offset_malloc.h"

    double * dev_dy_hx; //similar to hx in cpu code
    cudaStatus = hipMalloc((void **) &dev_dy_hx, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hx, &hx, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMemcpy failed!\n"); return;}

    double * dev_dy_hy;
    cudaStatus = hipMalloc((void **) &dev_dy_hy, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hy, &hy, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMemcpy failed!\n"); return;}

    double * dev_dy_hz;
    cudaStatus = hipMalloc((void **) &dev_dy_hz, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hz, &hz, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMemcpy failed!\n"); return;}

    int * dev_sz;
    cudaStatus = hipMalloc((void **) &dev_sz, 3*sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_sz, sz, 3*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMemcpy failed!\n"); return;}

    int * dev_zero;
    cudaStatus = hipMalloc((void **) &dev_zero, sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "0 hipMalloc failed!\n"); return;}

    double * dev_pmin;
    cudaStatus = hipMalloc((void **) &dev_pmin, 3*sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmin hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_pmin, pmin, 3*sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmin hipMemcpy failed!\n"); return;}

    double *dev_pmax;
    cudaStatus = hipMalloc((void **) &dev_pmax, sizeof(pmax)*sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmax hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_pmax, pmax, sizeof(pmax)*sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmax hipMemcpy failed!\n"); return;}

    bssn::timer::t_mem_handling_gpu.stop();

    // Allocate memory to store the output of derivs
    unsigned int n = sz[0]*sz[1]*sz[2];
    int size = n * sizeof(double);

    bssn::timer::t_deriv_gpu.start();

    bssn::timer::t_mem_handling_gpu.start();
    #include "bssnrhs_cuda_malloc.h"
    #include "bssnrhs_cuda_malloc_adv.h"
    bssn::timer::t_mem_handling_gpu.stop();
    
    // Deriv calls are follows
    cuda_calc_all(dev_var_in, dev_dy_hx,dev_dy_hy,dev_dy_hz,dev_zero, dev_sz, dev_bflag, sz,
#include "list_of_args.h"
    );

    #include "bssnrhs_cuda_derivs_adv.h"

    bssn::timer::t_deriv_gpu.stop();


    bssn::timer::t_rhs_gpu.start();
    calc_bssn_eqns(sz, dev_sz, dev_pmin, dev_dy_hz, dev_dy_hy, dev_dy_hx, dev_var_in, dev_var_out,
        #include "list_of_args.h"
    );
    bssn::timer::t_rhs_gpu.stop();

        
    if (bflag != 0) {
        bssn::timer::t_bdyc_gpu.start();

        bssn_bcs(dev_var_out, dev_var_in, dev_alphaInt, grad_0_alpha, grad_1_alpha, grad_2_alpha,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_chiInt, grad_0_chi, grad_1_chi, grad_2_chi,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_KInt, grad_0_K, grad_1_K, grad_2_K,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);

        bssn_bcs(dev_var_out, dev_var_in, dev_beta0Int, grad_0_beta0, grad_1_beta0, grad_2_beta0,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_beta1Int, grad_0_beta1, grad_1_beta1, grad_2_beta1,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_beta2Int, grad_0_beta2, grad_1_beta2, grad_2_beta2,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);

        bssn_bcs(dev_var_out, dev_var_in, dev_Gt0Int, grad_0_Gt0, grad_1_Gt0, grad_2_Gt0,
            dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_Gt1Int, grad_0_Gt1, grad_1_Gt1, grad_2_Gt1,
            dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_Gt2Int, grad_0_Gt2, grad_1_Gt2, grad_2_Gt2,
            dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz);

        bssn_bcs(dev_var_out, dev_var_in, dev_B0Int, grad_0_B0, grad_1_B0, grad_2_B0,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_B1Int, grad_0_B1, grad_1_B1, grad_2_B1,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_B2Int, grad_0_B2, grad_1_B2, grad_2_B2,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);

        bssn_bcs(dev_var_out, dev_var_in, dev_At0Int, grad_0_At0, grad_1_At0, grad_2_At0,
            dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At1Int, grad_0_At1, grad_1_At1, grad_2_At1,
            dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At2Int, grad_0_At2, grad_1_At2, grad_2_At2,
            dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At3Int, grad_0_At3, grad_1_At3, grad_2_At3,
            dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At4Int, grad_0_At4, grad_1_At4, grad_2_At4,
            dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At5Int, grad_0_At5, grad_1_At5, grad_2_At5,
            dev_pmin, dev_pmax, 2.0, 0.0, sz, dev_bflag, dev_sz); 

        bssn_bcs(dev_var_out, dev_var_in, dev_gt0Int, grad_0_gt0, grad_1_gt0, grad_2_gt0,
                dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt1Int, grad_0_gt1, grad_1_gt1, grad_2_gt1,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt2Int, grad_0_gt2, grad_1_gt2, grad_2_gt2,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt3Int, grad_0_gt3, grad_1_gt3, grad_2_gt3,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt4Int, grad_0_gt4, grad_1_gt4, grad_2_gt4,
            dev_pmin, dev_pmax, 1.0, 0.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt5Int, grad_0_gt5, grad_1_gt5, grad_2_gt5,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
          

        bssn::timer::t_bdyc_gpu.stop();
    }

    bssn::timer::t_deriv_gpu.start();
    #include "bssnrhs_cuda_ko_derivs.h"
    bssn::timer::t_deriv_gpu.stop();

    #if test && 0
    // Copying specified array to CPU for testing purpose
    double * host_array_cpu = (double *) malloc(size);
    #include "test_GPU_derivs.h" // only one of both at a time
    #include "test_GPU_adv_derivs.h"
    free(host_array_cpu); 
    #endif

    bssn::timer::t_rhs_gpu.start();
    get_output(dev_var_out, dev_sz, sz,
        #include "list_of_args.h"
    );
    bssn::timer::t_rhs_gpu.stop();

    
    bssn::timer::t_mem_handling_gpu.start();

    bssn::timer::t_deriv_gpu.start();
    #include "bssnrhs_cuda_mdealloc.h"
    #include "bssnrhs_cuda_mdealloc_adv.h"
    bssn::timer::t_deriv_gpu.stop();

    #include "bssnrhs_cuda_offset_demalloc.h"
    hipFree(dev_dy_hx);
    hipFree(dev_dy_hy);
    hipFree(dev_dy_hz);
    hipFree(dev_sz);
    hipFree(dev_zero);
    hipFree(dev_pmin);
    hipFree(dev_pmax);

    bssn::timer::t_mem_handling_gpu.stop();
}

__global__ void cacl_bssn_bcs_x(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    double *pmin, double *pmax, const double f_falloff, const double f_asymptotic,
    int *dev_sz, int* dev_bflag) {

        int j = 3 + threadIdx.x + blockIdx.x * blockDim.x;
        int k = 3 + threadIdx.y + blockIdx.y * blockDim.y;
        int nx = dev_sz[0];
        int ny = dev_sz[1];
        int nz = dev_sz[2];

        if(j >= ny-3 || k >= nz-3) return;

        double inv_r;
        double hx = (pmax[0] - pmin[0]) / (nx - 1);
        double hy = (pmax[1] - pmin[1]) / (ny - 1);
        double hz = (pmax[2] - pmin[2]) / (nz - 1);
        double x, y, z;
        int pp;

        if (*dev_bflag & (1u<<OCT_DIR_LEFT)) {
            
            x = pmin[0] + 3*hx;
            z = pmin[2] + k*hz;
            y = pmin[1] + j*hy;
            pp = IDX(3,j,k);
            inv_r = 1.0 / sqrt(x*x + y*y + z*z);
   
            output[*dev_u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                          + y * dyf[pp]
                          + z * dzf[pp]
                          + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
          }
        
          if (*dev_bflag & (1u<<OCT_DIR_RIGHT)) {
             x = pmin[0] + (nx - 3)*hx;
             z = pmin[2] + k*hz;
             y = pmin[1] + j*hy;
             pp = IDX((nx - 3),j,k);
             inv_r = 1.0 / sqrt(x*x + y*y + z*z);
    
             output[*dev_u_offset + pp] = -  inv_r * (
                             x * dxf[pp]
                           + y * dyf[pp]
                           + z * dzf[pp]
                           + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
          }

}

__global__ void cacl_bssn_bcs_y(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    double *pmin, double *pmax, const double f_falloff, const double f_asymptotic,
    int *dev_sz, int* dev_bflag) {

        int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
        int k = 3 + threadIdx.y + blockIdx.y * blockDim.y;
        int nx = dev_sz[0];
        int ny = dev_sz[1];
        int nz = dev_sz[2];

        if(i >= nx-3 || k >= nz-3) return;

        double inv_r;
        double hx = (pmax[0] - pmin[0]) / (nx - 1);
        double hy = (pmax[1] - pmin[1]) / (ny - 1);
        double hz = (pmax[2] - pmin[2]) / (nz - 1);
        double x, y, z;
        int pp;

        if (*dev_bflag & (1u<<OCT_DIR_DOWN)) {
            
            y = pmin[1] + 3*hy;
            z = pmin[2] + k*hz;
            x = pmin[0] + i*hx;
            pp = IDX(i,3,k);
            inv_r = 1.0 / sqrt(x*x + y*y + z*z);
   
            output[*dev_u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                          + y * dyf[pp]
                          + z * dzf[pp]
                          + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
            
          }
        
          if (*dev_bflag & (1u<<OCT_DIR_UP)) {
             x = pmin[0] + i*hx;
             z = pmin[2] + k*hz;
             y = pmin[1] + (ny-3)*hy;
             pp = IDX(i,(ny - 3),k);
             inv_r = 1.0 / sqrt(x*x + y*y + z*z);
    
             output[*dev_u_offset + pp] = -  inv_r * (
                             x * dxf[pp]
                           + y * dyf[pp]
                           + z * dzf[pp]
                           + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
               
          }
}

__global__ void cacl_bssn_bcs_z(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    double *pmin, double *pmax, const double f_falloff, const double f_asymptotic,
    int *dev_sz, int* dev_bflag) {
        
            int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
            int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
            int nx = dev_sz[0];
            int ny = dev_sz[1];
            int nz = dev_sz[2];

            if(i >= nx-3 || j >= ny-3) return;

            double inv_r;
            double hx = (pmax[0] - pmin[0]) / (nx - 1);
            double hy = (pmax[1] - pmin[1]) / (ny - 1);
            double hz = (pmax[2] - pmin[2]) / (nz - 1);
            double x, y, z;
            int pp;

            if (*dev_bflag & (1u<<OCT_DIR_BACK)) {
            
            y = pmin[1] + j*hy;
            z = pmin[2] + 3*hz;
            x = pmin[0] + i*hx;
            pp = IDX(i,j,3);
            inv_r = 1.0 / sqrt(x*x + y*y + z*z);
   
            output[*dev_u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                          + y * dyf[pp]
                          + z * dzf[pp]
                          + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
            
          }
        
          if (*dev_bflag & (1u<<OCT_DIR_FRONT)) {
            x = pmin[0] + i*hx;
            z = pmin[2] + (nz-3)*hz;
            y = pmin[1] + j*hy;
            pp = IDX(i,j,3);
            inv_r = 1.0 / sqrt(x*x + y*y + z*z);
    
            output[*dev_u_offset + pp] = -  inv_r * (
                             x * dxf[pp]
                           + y * dyf[pp]
                           + z * dzf[pp]
                           + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
               
          }
}

void bssn_bcs(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    double *pmin, double *pmax, const double f_falloff, const double f_asymptotic,
    const unsigned int *host_sz, int* dev_bflag, int* dev_sz) {
        
        hipError_t cudaStatus;
        const unsigned int nx = host_sz[0];
        const unsigned int ny = host_sz[1];
        const unsigned int nz = host_sz[2];

        const int ie = nx - 3;//x direction
        const int je = ny - 3;//y direction
        const int ke = nz - 3;//z direction

        int maximumIterations = (je>ke) ? je: ke;
        
        int requiredBlocks = (9 + maximumIterations) / 10;
        
        int threads_y = (requiredBlocks-1+je) / requiredBlocks;
        int threads_z = (requiredBlocks-1+ke) / requiredBlocks;
        
        cacl_bssn_bcs_x <<< dim3(threads_y,threads_z), dim3(threads_y,threads_z) >>> (output, dev_var_in,
           dev_u_offset, dxf, dyf, dzf, pmin, pmax, f_falloff, f_asymptotic, dev_sz, dev_bflag );

        cudaStatus = hipDeviceSynchronize();
           if (cudaStatus != hipSuccess) {
                   fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bssn_bcs_x kernal!\n", cudaStatus);
                   return;
           }
        
        maximumIterations = (ke>ie) ? ke : ie ;
        requiredBlocks = (9 + maximumIterations)/10;
        int threads_x = (requiredBlocks-1+ie) / requiredBlocks;
        threads_z = (requiredBlocks-1+ke) / requiredBlocks;
        cacl_bssn_bcs_y <<< dim3(threads_x,threads_z), dim3(threads_x,threads_z) >>> (output, dev_var_in,
            dev_u_offset, dxf, dyf, dzf, pmin, pmax, f_falloff, f_asymptotic, dev_sz, dev_bflag );
 
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bssn_bcs_y kernal!\n", cudaStatus);
            return;
        }

        maximumIterations = (je>ie) ? je : ie ;
        requiredBlocks = (9 + maximumIterations)/10;
        threads_x = (requiredBlocks-1+ie) / requiredBlocks;
        threads_y = (requiredBlocks-1+je) / requiredBlocks;
        cacl_bssn_bcs_z <<< dim3(threads_x,threads_y), dim3(threads_x,threads_y) >>> (output, dev_var_in,
            dev_u_offset, dxf, dyf, dzf, pmin, pmax, f_falloff, f_asymptotic, dev_sz, dev_bflag );
 
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bssn_bcs_z kernal!\n", cudaStatus);
            return;
        }
    }

    __global__ void kernal_get_output (double * output, int * dev_sz, 
        #include "list_of_para.h"
    ) 
    {

        int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
        int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
        int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

        int nx = dev_sz[0];
        int ny = dev_sz[1];

        if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

        const  double sigma = 1e-4;
        int pp = i + nx*(j + ny*k);

        output[*dev_alphaInt + pp] += sigma * (grad_0_alpha[pp] + grad_1_alpha[pp] + grad_2_alpha[pp]);
        output[*dev_beta0Int + pp] += sigma * (grad_0_beta0[pp] + grad_1_beta0[pp] + grad_2_beta0[pp]);
        output[*dev_beta1Int + pp] += sigma * (grad_0_beta1[pp] + grad_1_beta1[pp] + grad_2_beta1[pp]);
        output[*dev_beta2Int + pp] += sigma * (grad_0_beta2[pp] + grad_1_beta2[pp] + grad_2_beta2[pp]);

        output[*dev_gt0Int + pp] += sigma * (grad_0_gt0[pp] + grad_1_gt0[pp] + grad_2_gt0[pp]);
        output[*dev_gt1Int + pp] += sigma * (grad_0_gt1[pp] + grad_1_gt1[pp] + grad_2_gt1[pp]);
        output[*dev_gt2Int + pp] += sigma * (grad_0_gt2[pp] + grad_1_gt2[pp] + grad_2_gt2[pp]);
        output[*dev_gt3Int + pp] += sigma * (grad_0_gt3[pp] + grad_1_gt3[pp] + grad_2_gt3[pp]);
        output[*dev_gt4Int + pp] += sigma * (grad_0_gt4[pp] + grad_1_gt4[pp] + grad_2_gt4[pp]);
        output[*dev_gt5Int + pp] += sigma * (grad_0_gt5[pp] + grad_1_gt5[pp] + grad_2_gt5[pp]);

        output[*dev_chiInt + pp]  += sigma * (grad_0_chi[pp] + grad_1_chi[pp] + grad_2_chi[pp]);

        output[*dev_At0Int + pp] += sigma * (grad_0_At0[pp] + grad_1_At0[pp] + grad_2_At0[pp]);
        output[*dev_At1Int + pp] += sigma * (grad_0_At1[pp] + grad_1_At1[pp] + grad_2_At1[pp]);
        output[*dev_At2Int + pp] += sigma * (grad_0_At2[pp] + grad_1_At2[pp] + grad_2_At2[pp]);
        output[*dev_At3Int + pp] += sigma * (grad_0_At3[pp] + grad_1_At3[pp] + grad_2_At3[pp]);
        output[*dev_At4Int + pp] += sigma * (grad_0_At4[pp] + grad_1_At4[pp] + grad_2_At4[pp]);
        output[*dev_At5Int + pp] += sigma * (grad_0_At5[pp] + grad_1_At5[pp] + grad_2_At5[pp]);

        output[*dev_KInt + pp] += sigma * (grad_0_K[pp] + grad_1_K[pp] + grad_2_K[pp]);
        
        output[*dev_Gt0Int + pp] += sigma * (grad_0_Gt0[pp] + grad_1_Gt0[pp] + grad_2_Gt0[pp]);
        output[*dev_Gt1Int + pp] += sigma * (grad_0_Gt1[pp] + grad_1_Gt1[pp] + grad_2_Gt1[pp]);
        output[*dev_Gt2Int + pp] += sigma * (grad_0_Gt2[pp] + grad_1_Gt2[pp] + grad_2_Gt2[pp]);

        output[*dev_B0Int + pp] += sigma * (grad_0_B0[pp] + grad_1_B0[pp] + grad_2_B0[pp]);
        output[*dev_B1Int + pp] += sigma * (grad_0_B1[pp] + grad_1_B1[pp] + grad_2_B1[pp]);
        output[*dev_B2Int + pp] += sigma * (grad_0_B2[pp] + grad_1_B2[pp] + grad_2_B2[pp]);

    }

    void get_output (double* output, int* dev_sz, const unsigned int* host_sz, 
        #include "list_of_para.h"
    ) 
    {
            const int ie = host_sz[0] - 3;//x direction
            const int je = host_sz[1] - 3;//y direction
            const int ke = host_sz[2] - 3;//z direction
  
            int temp_max = (ie>je)? ie : je;
            int maximumIterations = (temp_max>ke) ? temp_max: ke;
            
            int requiredBlocks = (9+maximumIterations) / 10;

            kernal_get_output <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                      dim3((ie + requiredBlocks -1)/requiredBlocks,
                      (je + requiredBlocks -1)/requiredBlocks, 
                      (ke + requiredBlocks -1)/requiredBlocks) >>> (output, dev_sz, 
                        #include "list_of_args.h"
                      );
    }
