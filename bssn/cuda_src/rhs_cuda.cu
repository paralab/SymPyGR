#include "hip/hip_runtime.h"
#include "rhs_cuda.h"
#include "bssneqn_solve.h"

#include <iostream>
#include <stdio.h>


enum VAR_CU {U_ALPHA=0,U_CHI,U_K,U_GT0,U_GT1,U_GT2,U_BETA0,U_BETA1,U_BETA2,U_B0,U_B1,U_B2,U_SYMGT0,U_SYMGT1,U_SYMGT2,U_SYMGT3,U_SYMGT4,U_SYMGT5,U_SYMAT0,U_SYMAT1,U_SYMAT2,U_SYMAT3,U_SYMAT4,U_SYMAT5};

void cuda_bssnrhs(double * dev_var_out, double * dev_var_in, const unsigned int unzip_dof, 
const unsigned int& offset, const double *pmin, const double *pmax, const unsigned int *sz, 
const unsigned int& bflag)
{ 
    int alphaInt = (VAR_CU::U_ALPHA) * unzip_dof + offset;
    int chiInt = (VAR_CU::U_CHI) * unzip_dof + offset;
    int KInt = (VAR_CU::U_K) * unzip_dof + offset;
    int gt0Int = (VAR_CU::U_SYMGT0) * unzip_dof + offset;
    int gt1Int = (VAR_CU::U_SYMGT1) * unzip_dof + offset;
    int gt2Int =  (VAR_CU::U_SYMGT2) * unzip_dof + offset;
    int gt3Int =(VAR_CU::U_SYMGT3) * unzip_dof + offset;
    int gt4Int = (VAR_CU::U_SYMGT4) * unzip_dof + offset;
    int gt5Int = (VAR_CU::U_SYMGT5) * unzip_dof + offset;
    int beta0Int = (VAR_CU::U_BETA0) * unzip_dof + offset;
    int beta1Int = (VAR_CU::U_BETA1) * unzip_dof + offset;
    int beta2Int =(VAR_CU::U_BETA2) * unzip_dof + offset;
    int At0Int = (VAR_CU::U_SYMAT0) * unzip_dof + offset;
    int At1Int = (VAR_CU::U_SYMAT1) * unzip_dof + offset;
    int At2Int = (VAR_CU::U_SYMAT2) * unzip_dof + offset;
    int At3Int = (VAR_CU::U_SYMAT3) * unzip_dof + offset;
    int At4Int = (VAR_CU::U_SYMAT4) * unzip_dof + offset;
    int At5Int = (VAR_CU::U_SYMAT5) * unzip_dof + offset;
    int Gt0Int = (VAR_CU::U_GT0) * unzip_dof + offset;
    int Gt1Int = (VAR_CU::U_GT1) * unzip_dof + offset;
    int Gt2Int = (VAR_CU::U_GT2) * unzip_dof + offset;
    int B0Int = (VAR_CU::U_B0) * unzip_dof + offset;
    int B1Int = (VAR_CU::U_B1) * unzip_dof + offset;
    int B2Int = (VAR_CU::U_B2) * unzip_dof + offset;

    double hx = (pmax[0] - pmin[0]) / (sz[0] - 1);
    double hy = (pmax[1] - pmin[1]) / (sz[1] - 1);
    double hz = (pmax[2] - pmin[2]) / (sz[2] - 1);

    // Send above values to GPU memory
    hipError_t cudaStatus;
    #include "bssnrhs_cuda_offset_malloc.h"

    double * dev_dy_hx; //similar to hx in cpu code
    cudaStatus = hipMalloc((void **) &dev_dy_hx, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hx, &hx, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hx hipMemcpy failed!\n"); return;}

    double * dev_dy_hy;
    cudaStatus = hipMalloc((void **) &dev_dy_hy, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hy, &hy, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hy hipMemcpy failed!\n"); return;}

    double * dev_dy_hz;
    cudaStatus = hipMalloc((void **) &dev_dy_hz, sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_dy_hz, &hz, sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "hz hipMemcpy failed!\n"); return;}

    int * dev_sz;
    cudaStatus = hipMalloc((void **) &dev_sz, 3*sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_sz, sz, 3*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "sz hipMemcpy failed!\n"); return;}

    int * dev_zero;
    cudaStatus = hipMalloc((void **) &dev_zero, sizeof(int));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "0 hipMalloc failed!\n"); return;}

    double * dev_pmin;
    cudaStatus = hipMalloc((void **) &dev_pmin, 3*sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmin hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_pmin, pmin, 3*sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmin hipMemcpy failed!\n"); return;}

    double *dev_pmax;
    cudaStatus = hipMalloc((void **) &dev_pmax, sizeof(pmax)*sizeof(double));
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmax hipMalloc failed!\n"); return;}
    cudaStatus = hipMemcpy(dev_pmax, pmax, sizeof(pmax)*sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {fprintf(stderr, "pmax hipMemcpy failed!\n"); return;}

    // Allocate memory to store the output of derivs
    unsigned int n = sz[0]*sz[1]*sz[2];
    int size = n * sizeof(double);

    bssn::timer::t_deriv_gpu.start();

    #include "bssnrhs_cuda_malloc.h"
    #include "bssnrhs_cuda_malloc_adv.h"
   

    // Deriv calls are follows
    #include "bssnrhs_cuda_derivs.h"
    #include "bssnrhs_cuda_derivs_adv.h"

    bssn::timer::t_deriv_gpu.stop();


    bssn::timer::t_rhs_gpu.start();
    calc_bssn_eqns(sz, dev_sz, dev_pmin, dev_dy_hz, dev_dy_hy, dev_dy_hx, dev_var_in, dev_var_out,
        #include "list_of_args.h"
    );
    bssn::timer::t_rhs_gpu.stop();
    #if !testUntilBssnEqs
    #if test
    // // Copying specified array to CPU for testing purpose
    // double * host_array_cpu = (double *) malloc(size);
    // cudaStatus = hipMemcpy(host_array_cpu, agrad_0_alpha, size, hipMemcpyDeviceToHost);
    // if (cudaStatus != hipSuccess) {fprintf(stderr, "TEST: host_array_cpu hipMemcpy from GPU to CPU failed!\n"); return;}
    // test_file_write::writeToFile("output_cuda.txt", host_array_cpu, n);
    // free(host_array_cpu);
    #endif
        
    if (bflag != 0) {
        bssn::timer::t_bdyc_gpu.start();

        bssn_bcs(dev_var_out, dev_var_in, dev_alphaInt, grad_0_alpha, grad_1_alpha, grad_2_alpha,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_chiInt, grad_0_chi, grad_1_chi, grad_2_chi,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_KInt, grad_0_K, grad_1_K, grad_2_K,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_beta0Int, grad_0_beta0, grad_1_beta0, grad_2_beta0,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_beta1Int, grad_0_beta1, grad_1_beta1, grad_2_beta1,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_beta2Int, grad_0_beta2, grad_1_beta2, grad_2_beta2,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_Gt0Int, grad_0_Gt0, grad_1_Gt0, grad_2_Gt0,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_Gt1Int, grad_0_Gt1, grad_1_Gt1, grad_2_Gt1,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_Gt2Int, grad_0_Gt2, grad_1_Gt2, grad_2_Gt2,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_B0Int, grad_0_B0, grad_1_B0, grad_2_B0,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_B1Int, grad_0_B1, grad_1_B1, grad_2_B1,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_B2Int, grad_0_B2, grad_1_B2, grad_2_B2,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At0Int, grad_0_At0, grad_1_At0, grad_2_At0,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At1Int, grad_0_At1, grad_1_At1, grad_2_At1,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At2Int, grad_0_At2, grad_1_At2, grad_2_At2,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At3Int, grad_0_At3, grad_1_At3, grad_2_At3,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At4Int, grad_0_At4, grad_1_At4, grad_2_At4,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_At5Int, grad_0_At5, grad_1_At5, grad_2_At5,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);    
        bssn_bcs(dev_var_out, dev_var_in, dev_gt0Int, grad_0_gt0, grad_1_gt0, grad_2_gt0,
                dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt1Int, grad_0_gt1, grad_1_gt1, grad_2_gt1,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt2Int, grad_0_gt2, grad_1_gt2, grad_2_gt2,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt3Int, grad_0_gt3, grad_1_gt3, grad_2_gt3,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt4Int, grad_0_gt4, grad_1_gt4, grad_2_gt4,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
        bssn_bcs(dev_var_out, dev_var_in, dev_gt5Int, grad_0_gt5, grad_1_gt5, grad_2_gt5,
            dev_pmin, dev_pmax, 1.0, 1.0, sz, dev_bflag, dev_sz);
          

        bssn::timer::t_bdyc_gpu.stop();
    }

    bssn::timer::t_deriv_gpu.start();
    #include "bssnrhs_cuda_ko_derivs.h"
    bssn::timer::t_deriv_gpu.stop();

    bssn::timer::t_rhs_gpu.start();
    get_output(dev_var_out, dev_sz, sz,
        #include "list_of_args.h"
    );
    bssn::timer::t_rhs_gpu.stop();

    bssn::timer::t_deriv_gpu.start();
    #include "bssnrhs_cuda_offset_demalloc.h"
    #include "bssnrhs_cuda_mdealloc.h"
    #include "bssnrhs_cuda_mdealloc_adv.h"
    hipFree(dev_dy_hx);
    hipFree(dev_dy_hy);
    hipFree(dev_dy_hz);
    hipFree(dev_sz);
    hipFree(dev_zero);
    hipFree(dev_pmin);
    bssn::timer::t_deriv_gpu.stop();

    #endif
}

__global__ void cacl_bssn_bcs_x(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    double *pmin, double *pmax, const double f_falloff, const double f_asymptotic,
    int *dev_sz, int* dev_bflag) {

        int j = 3 + threadIdx.x + blockIdx.x * blockDim.x;
        int k = 3 + threadIdx.y + blockIdx.y * blockDim.y;
        int nx = dev_sz[0];
        int ny = dev_sz[1];
        int nz = dev_sz[2];

        if(j >= ny-3 || k >= nz-3) return;

        double inv_r;
        double hx = (pmax[0] - pmin[0]) / (nx - 1);
        double hy = (pmax[1] - pmin[1]) / (ny - 1);
        double hz = (pmax[2] - pmin[2]) / (nz - 1);
        int x, y, z, pp;

        if (*dev_bflag & (1u<<OCT_DIR_LEFT)) {
            
            x = pmin[0] + 3*hx;
            z = pmin[2] + k*hz;
            y = pmin[1] + j*hy;
            pp = IDX(3,j,k);
            inv_r = 1.0 / sqrt((float)(x*x + y*y + z*z));
   
            output[*dev_u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                          + y * dyf[pp]
                          + z * dzf[pp]
                          + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
            
          }
        
          if (*dev_bflag & (1u<<OCT_DIR_RIGHT)) {
             x = pmin[0] + (nx - 3)*hx;
             z = pmin[2] + k*hz;
             y = pmin[1] + j*hy;
             pp = IDX((nx - 3),j,k);
             inv_r = 1.0 / sqrt((float)(x*x + y*y + z*z));
    
             output[*dev_u_offset + pp] = -  inv_r * (
                             x * dxf[pp]
                           + y * dyf[pp]
                           + z * dzf[pp]
                           + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
               
          }

}

__global__ void cacl_bssn_bcs_y(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    double *pmin, double *pmax, const double f_falloff, const double f_asymptotic,
    int *dev_sz, int* dev_bflag) {

        int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
        int k = 3 + threadIdx.y + blockIdx.y * blockDim.y;
        int nx = dev_sz[0];
        int ny = dev_sz[1];
        int nz = dev_sz[2];

        if(i >= nx-3 || k >= nz-3) return;

        double inv_r;
        double hx = (pmax[0] - pmin[0]) / (nx - 1);
        double hy = (pmax[1] - pmin[1]) / (ny - 1);
        double hz = (pmax[2] - pmin[2]) / (nz - 1);
        int x, y, z, pp;

        if (*dev_bflag & (1u<<OCT_DIR_DOWN)) {
            
            y = pmin[1] + 3*hy;
            z = pmin[2] + k*hz;
            x = pmin[0] + i*hx;
            pp = IDX(i,3,k);
            inv_r = 1.0 / sqrt((float)(x*x + y*y + z*z));
   
            output[*dev_u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                          + y * dyf[pp]
                          + z * dzf[pp]
                          + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
            
          }
        
          if (*dev_bflag & (1u<<OCT_DIR_UP)) {
             x = pmin[0] + i*hx;
             z = pmin[2] + k*hz;
             y = pmin[1] + (ny-3)*hy;
             pp = IDX(i,(ny - 3),k);
             inv_r = 1.0 / sqrt((float)(x*x + y*y + z*z));
    
             output[*dev_u_offset + pp] = -  inv_r * (
                             x * dxf[pp]
                           + y * dyf[pp]
                           + z * dzf[pp]
                           + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
               
          }
}

__global__ void cacl_bssn_bcs_z(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    double *pmin, double *pmax, const double f_falloff, const double f_asymptotic,
    int *dev_sz, int* dev_bflag) {
        
            int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
            int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
            int nx = dev_sz[0];
            int ny = dev_sz[1];
            int nz = dev_sz[2];

            if(i >= nx-3 || j >= ny-3) return;

            double inv_r;
            double hx = (pmax[0] - pmin[0]) / (nx - 1);
            double hy = (pmax[1] - pmin[1]) / (ny - 1);
            double hz = (pmax[2] - pmin[2]) / (nz - 1);
            int x, y, z, pp;

            if (*dev_bflag & (1u<<OCT_DIR_BACK)) {
            
            y = pmin[1] + j*hy;
            z = pmin[2] + 3*hz;
            x = pmin[0] + i*hx;
            pp = IDX(i,j,3);
            inv_r = 1.0 / sqrt((float)(x*x + y*y + z*z));
   
            output[*dev_u_offset + pp] = -  inv_r * (
                            x * dxf[pp]
                          + y * dyf[pp]
                          + z * dzf[pp]
                          + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
            
          }
        
          if (*dev_bflag & (1u<<OCT_DIR_FRONT)) {
            x = pmin[0] + i*hx;
            z = pmin[2] + (nz-3)*hz;
            y = pmin[1] + j*hy;
            pp = IDX(i,j,3);
            inv_r = 1.0 / sqrt((float)(x*x + y*y + z*z));
    
            output[*dev_u_offset + pp] = -  inv_r * (
                             x * dxf[pp]
                           + y * dyf[pp]
                           + z * dzf[pp]
                           + f_falloff * (   dev_var_in[*dev_u_offset + pp] - f_asymptotic ) );
               
          }
}

void bssn_bcs(double * output, double * dev_var_in, int* dev_u_offset,
    double *dxf, double *dyf, double *dzf,
    double *pmin, double *pmax, const double f_falloff, const double f_asymptotic,
    const unsigned int *host_sz, int* dev_bflag, int* dev_sz) {
        
        hipError_t cudaStatus;
        const unsigned int nx = host_sz[0];
        const unsigned int ny = host_sz[1];
        const unsigned int nz = host_sz[2];

        const int ie = nx - 3;//x direction
        const int je = ny - 3;//y direction
        const int ke = nz - 3;//z direction

        int maximumIterations = (je>ke) ? je: ke;
        
        int requiredBlocks = (9 + maximumIterations) / 10;
        
        int threads_y = (requiredBlocks-1+je) / requiredBlocks;
        int threads_z = (requiredBlocks-1+ke) / requiredBlocks;
        
        cacl_bssn_bcs_x <<< dim3(threads_y,threads_z), dim3(threads_y,threads_z) >>> (output, dev_var_in,
           dev_u_offset, dxf, dyf, dzf, pmin, pmax, f_falloff, f_asymptotic, dev_sz, dev_bflag );

        cudaStatus = hipDeviceSynchronize();
           if (cudaStatus != hipSuccess) {
                   fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bssn_bcs_x kernal!\n", cudaStatus);
                   return;
           }
        
        maximumIterations = (ke>ie) ? ke : ie ;
        requiredBlocks = (9 + maximumIterations)/10;
        int threads_x = (requiredBlocks-1+ie) / requiredBlocks;
        threads_z = (requiredBlocks-1+ke) / requiredBlocks;
        cacl_bssn_bcs_y <<< dim3(threads_x,threads_z), dim3(threads_x,threads_z) >>> (output, dev_var_in,
            dev_u_offset, dxf, dyf, dzf, pmin, pmax, f_falloff, f_asymptotic, dev_sz, dev_bflag );
 
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bssn_bcs_y kernal!\n", cudaStatus);
            return;
        }

        maximumIterations = (je>ie) ? je : ie ;
        requiredBlocks = (9 + maximumIterations)/10;
        threads_x = (requiredBlocks-1+ie) / requiredBlocks;
        threads_y = (requiredBlocks-1+je) / requiredBlocks;
        cacl_bssn_bcs_z <<< dim3(threads_x,threads_y), dim3(threads_x,threads_y) >>> (output, dev_var_in,
            dev_u_offset, dxf, dyf, dzf, pmin, pmax, f_falloff, f_asymptotic, dev_sz, dev_bflag );
 
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bssn_bcs_z kernal!\n", cudaStatus);
            return;
        }
    }

    __global__ void kernal_get_output (double * output, int * dev_sz, 
        #include "list_of_para.h"
    ) 
    {

        int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
        int j = 1 + threadIdx.y + blockIdx.y * blockDim.y;
        int k = 1 + threadIdx.z + blockIdx.z * blockDim.z;

        int nx = dev_sz[0];
        int ny = dev_sz[1];

        if(i >= nx-3 || j >= ny-1 || k >= dev_sz[2]-1) return;

        const  double sigma = 1e-4;
        int pp = i + nx*(j + ny*k);

        output[*dev_alphaInt + pp] += sigma * (grad_0_alpha[pp] + grad_1_alpha[pp] + grad_2_alpha[pp]);
        output[*dev_beta0Int + pp] += sigma * (grad_0_beta0[pp] + grad_1_beta0[pp] + grad_2_beta0[pp]);
        output[*dev_beta1Int + pp] += sigma * (grad_0_beta1[pp] + grad_1_beta1[pp] + grad_2_beta1[pp]);
        output[*dev_beta2Int + pp] += sigma * (grad_0_beta2[pp] + grad_1_beta2[pp] + grad_2_beta2[pp]);

        output[*dev_gt0Int + pp] += sigma * (grad_0_gt0[pp] + grad_1_gt0[pp] + grad_2_gt0[pp]);
        output[*dev_gt1Int + pp] += sigma * (grad_0_gt1[pp] + grad_1_gt1[pp] + grad_2_gt1[pp]);
        output[*dev_gt2Int + pp] += sigma * (grad_0_gt2[pp] + grad_1_gt2[pp] + grad_2_gt2[pp]);
        output[*dev_gt3Int + pp] += sigma * (grad_0_gt3[pp] + grad_1_gt3[pp] + grad_2_gt3[pp]);
        output[*dev_gt4Int + pp] += sigma * (grad_0_gt4[pp] + grad_1_gt4[pp] + grad_2_gt4[pp]);
        output[*dev_gt5Int + pp] += sigma * (grad_0_gt5[pp] + grad_1_gt5[pp] + grad_2_gt5[pp]);

        output[*dev_chiInt + pp]  += sigma * (grad_0_chi[pp] + grad_1_chi[pp] + grad_2_chi[pp]);

        output[*dev_At0Int + pp] += sigma * (grad_0_At0[pp] + grad_1_At0[pp] + grad_2_At0[pp]);
        output[*dev_At1Int + pp] += sigma * (grad_0_At1[pp] + grad_1_At1[pp] + grad_2_At1[pp]);
        output[*dev_At2Int + pp] += sigma * (grad_0_At2[pp] + grad_1_At2[pp] + grad_2_At2[pp]);
        output[*dev_At3Int + pp] += sigma * (grad_0_At3[pp] + grad_1_At3[pp] + grad_2_At3[pp]);
        output[*dev_At4Int + pp] += sigma * (grad_0_At4[pp] + grad_1_At4[pp] + grad_2_At4[pp]);
        output[*dev_At5Int + pp] += sigma * (grad_0_At5[pp] + grad_1_At5[pp] + grad_2_At5[pp]);

        output[*dev_KInt + pp] += sigma * (grad_0_K[pp] + grad_1_K[pp] + grad_2_K[pp]);

        output[*dev_Gt0Int + pp] += sigma * (grad_0_Gt0[pp] + grad_1_Gt0[pp] + grad_2_Gt0[pp]);
        output[*dev_Gt1Int + pp] += sigma * (grad_0_Gt1[pp] + grad_1_Gt1[pp] + grad_2_Gt1[pp]);
        output[*dev_Gt2Int + pp] += sigma * (grad_0_Gt2[pp] + grad_1_Gt2[pp] + grad_2_Gt2[pp]);

        output[*dev_B0Int + pp] += sigma * (grad_0_B0[pp] + grad_1_B0[pp] + grad_2_B0[pp]);
        output[*dev_B1Int + pp] += sigma * (grad_0_B1[pp] + grad_1_B1[pp] + grad_2_B1[pp]);
        output[*dev_B2Int + pp] += sigma * (grad_0_B2[pp] + grad_1_B2[pp] + grad_2_B2[pp]);

    }

    void get_output (double* output, int* dev_sz, const unsigned int* host_sz, 
        #include "list_of_para.h"
    ) 
    {
            const int ie = host_sz[0] - 3;//x direction
            const int je = host_sz[1] - 1;//y direction
            const int ke = host_sz[2] - 1;//z direction
  
            int temp_max = (ie>je)? ie : je;
            int maximumIterations = (temp_max>ke) ? temp_max: ke;
            
            int requiredBlocks = (9+maximumIterations) / 10;

            kernal_get_output <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                      dim3((ie + requiredBlocks -1)/requiredBlocks,
                      (je + requiredBlocks -1)/requiredBlocks, 
                      (ke + requiredBlocks -1)/requiredBlocks) >>> (output, dev_sz, 
                        #include "list_of_args.h"
                      );
    }
