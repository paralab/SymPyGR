#include "hip/hip_runtime.h"
/**
 * kernal.cu
 * 
 * Created on: Feb 12, 2018
 * 		Author: Akila
 **/

 #include "derivs_cuda.h"
 #include "rhs.h"
 #include "hip/hip_runtime.h"
 #include ""
 #include <stdio.h>
 
 __global__ void firstThreeForLoops(double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[1]-3-3)<=y ){ return; } else { j = x+3; } //j handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = x+3; } //k handler

    

    int nx = dev_sz[0];
    int ny = dev_sz[1];
    int pp = IDX(i, j, k);

    // printf("%f\n", (dev_var_in[(*dev_u_offset) + pp - 2*dev_sz[0]]));
    // double f = dev_var_in[(*dev_u_offset) + pp - 2*dev_sz[0]] - 8.0*dev_var_in[(*dev_u_offset) + pp - dev_sz[0]] + 8.0*dev_var_in[(*dev_u_offset) + pp + dev_sz[0]] - dev_var_in[(*dev_u_offset) + pp + 2*dev_sz[0]]; //*((1.0/dev_dy[0])/12.0);
    double f = (dev_var_in[(*dev_u_offset) + pp - 2*dev_sz[0]] - 8.0*dev_var_in[(*dev_u_offset) + pp - dev_sz[0]] + 8.0*dev_var_in[(*dev_u_offset) + pp + dev_sz[0]] - dev_var_in[(*dev_u_offset) + pp + 2*dev_sz[0]] )*((1.0/dev_dy[0])/12.0);
    printf("%f\n", f);

 }
 
 void deriv42_yWithCuda(double * dev_var_in, int u_offset, double dy, const unsigned int *sz, unsigned bflag)
 {

    int * dev_sz;
    double * dev_dy;
    int * dev_u_offset;

    // std::cout << sz[0] << std::endl;

    hipMalloc((void **) &dev_dy, sizeof(double));
    hipMalloc((void **) &dev_sz, 3*sizeof(int));
    hipMalloc((void **) &dev_u_offset, sizeof(int));

    hipMemcpy(dev_dy, &dy, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_sz, sz, 3*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_u_offset, &u_offset, sizeof(int), hipMemcpyHostToDevice);

    int zblocks = ((sz[2]-1)/10)+1;
    int yblocks = ((sz[0]-3)/10)+1;
    int xblocks = ((sz[1]-3)/10)+1;
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // for (int k = kb; k < ke; k++) {
    //     for (int i = ib; i < ie; i++) {
    //       for (int j = jb; j < je; j++) {
    //         int pp = IDX(i,j,k); //(i) + nx * ( (j) + ny * (k) )
    //         Dyu[pp] = (u[pp-2*nx] - 8.0*u[pp-nx] + 8.0*u[pp+nx] - u[pp+2*nx])*idy_by_12;
    //         // printf("%f\n", u[0]);
    //       }
    //     }
    //   }
    
    //   if (bflag & (1u<<OCT_DIR_DOWN)) {
    //     for (int k = kb; k < ke; k++) {
    //       for (int i = ib; i < ie; i++) {
    //         Dyu[IDX(i, 3,k)] = ( - 3.0 * u[IDX(i,3,k)]
    //                             +  4.0 * u[IDX(i,4,k)]
    //                             -        u[IDX(i,5,k)]
    //                           ) * idy_by_2;
    
    //         Dyu[IDX(i,4,k)] = ( - u[IDX(i,3,k)]
    //                             + u[IDX(i,5,k)]
    //                           ) * idy_by_2;
    //       }
    //     }
    //   }
    
    //   if (bflag & (1u<<OCT_DIR_UP)) {
    //     for (int k = kb; k < ke; k++) {
    //       for (int i = ib; i < ie; i++) {
    //         Dyu[IDX(i,je-2,k)] = ( - u[IDX(i,je-3,k)]
    //                                + u[IDX(i,je-1,k)]
    //                              ) * idy_by_2;
    
    //         Dyu[IDX(i,je-1,k)] = (        u[IDX(i,je-3,k)]
    //                               - 4.0 * u[IDX(i,je-2,k)]
    //                               + 3.0 * u[IDX(i,je-1,k)]
    //                           ) * idy_by_2;
    //       }
    //     }
    //   } 
    hipFree(&dev_sz);
    hipFree(&dev_dy);
    hipFree(&dev_u_offset);
    
    // #ifdef DEBUG_DERIVS_COMP
    //   for (int k = 3; k < sz[2]-3; k++) {
    //     for (int j = 3; j < sz[1]-3; j++) {
    //       for (int i = 3; i < sz[0]-3; i++) {
    //         int pp = IDX(i,j,k);
    //         if(std::isnan(Dyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //       }
    //     }
    //   }
    // #endif
 }
 
 