#include "hip/hip_runtime.h"
/**
 * Created on: March 15, 2018
 * 		Author: Akila
 **/

 #include "derivs_cuda.h"

__global__ void cuda_deriv42_y_firstThreeForLoops(double* output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[1]-3-3)<=y ){ return; } else { j = y+3; } //j handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[(*dev_u_offset) + pp - 2*dev_sz[0]] - 8.0*dev_var_in[(*dev_u_offset) + pp - dev_sz[0]] + 8.0*dev_var_in[(*dev_u_offset) + pp + dev_sz[0]] - dev_var_in[(*dev_u_offset) + pp + 2*dev_sz[0]] )*((1.0/dev_dy[0])/12.0);
    // printf("%f\n", output[pp]);
}

__global__ void cuda_deriv42_y_secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
    int x = threadIdx.x + blockIdx.x*30;
    int z = threadIdx.y + blockIdx.x*30;

    int i;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0];
    int ny = dev_sz[1];

    int pp3 = IDX(i, 3, k);
    int pp4 = IDX(i, 4, k);
    int pp5 = IDX(i, 5, k);

    output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] +  4*dev_var_in[(*dev_u_offset) + pp4] - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
    output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) + pp3]) * (0.50/dev_dy[0]);
    // printf("%f\n", output[pp3]);
 }

 __global__ void cuda_deriv42_y_thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*30;
    int z = threadIdx.y + blockIdx.x*30;

    int i;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0];
    int ny = dev_sz[1];

    int pp2 = IDX(i, dev_sz[1]-5, k); // IDX(i,je-2,k)
    int pp3 = IDX(i, dev_sz[1]-6, k); // IDX(i,je-3,k)
    int pp1 = IDX(i, dev_sz[1]-4, k); // IDX(i,je-1,k)

    output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) * 0.50 / dev_dy[0];
    output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]+ 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];
    // printf("%f\n", output[pp1]);
 }
 
 // Please some one verify the below kernals carefully -------------------------------------------------------------------------------
 __global__ void cuda_deriv42_x_firstThreeForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[1]-1-1)<=y ){ return; } else { j = y+1; } //j handler
    if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[(*dev_u_offset) + pp - 2] - 8.0*dev_var_in[(*dev_u_offset) + pp - 1] + 8.0*dev_var_in[(*dev_u_offset) + pp + 1] - dev_var_in[(*dev_u_offset) + pp + 2] )*((1.0/dev_dy[0])/12.0);
    // printf("%f\n", output[pp]);
}

__global__ void cuda_deriv42_x_secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int j;
   int k;

   if( (dev_sz[1]-1-1)<=x ){ return; } else { j = x+1; } 
   if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp3 = IDX(3, j, k);
   int pp4 = IDX(4, j, k);
   int pp5 = IDX(5, j, k);

   output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] + 4*dev_var_in[(*dev_u_offset) + pp4] - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
   output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) + pp3]) * (0.50/dev_dy[0]);
   // printf("%f\n", output[pp3]);
}

__global__ void cuda_deriv42_x_thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int j;
   int k;

   if( (dev_sz[1]-1-1)<=x ){ return; } else { j = x+1; } 
   if( (dev_sz[2]-1-1)<=z ){ return; } else { k = z+1; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp2 = IDX(dev_sz[0]-5, j, k); // IDX(ie-2,j,k)
   int pp3 = IDX(dev_sz[0]-6, j, k); // IDX(ie-3,j,k)
   int pp1 = IDX(dev_sz[0]-4,j,k); // IDX(ie-1,j,k)

   output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) * 0.50 / dev_dy[0];
   output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]+ 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];
   // printf("%f\n", output[pp1]);
}

__global__ void cuda_deriv42_z_firstThreeForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*10;
   int y = threadIdx.y + blockIdx.x*10;
   int z = threadIdx.z + blockIdx.x*10;

   int i;
   int j;
   int k;

   if( (dev_sz[2]-3-3)<=x ){ return; } else { k = x+3; }
   if( (dev_sz[0]-3-3)<=y ){ return; } else { i = y+3; }
   if( (dev_sz[1]-3-3)<=z ){ return; } else { j = z+3; }

   int nx = dev_sz[0]; 
   int ny = dev_sz[1]; 
   int n = nx * ny;
   int pp = IDX(i, j, k);
   
   output[pp] = (dev_var_in[(*dev_u_offset) + pp - 2*n] - 8.0*dev_var_in[(*dev_u_offset) + pp - n] + 8.0*dev_var_in[(*dev_u_offset) + pp + n] - dev_var_in[(*dev_u_offset) + pp + 2*n]) * ((1.0/dev_dy[0])/12);
   // printf("%f\n", output[pp]);
}

__global__ void cuda_deriv42_z_secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int j;
   int i;

   if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } 
   if( (dev_sz[1]-3-3)<=z ){ return; } else { j = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp3 = IDX(i, j, 3); // IDX(i, j, 3)
   int pp4 = IDX(i, j, 4); // IDX(i,j,4)
   int pp5 = IDX(i, j, 5); // IDX(i,j,5)

   output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] + 4*dev_var_in[(*dev_u_offset) + pp4] - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
   output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) + pp3]) * (0.50/dev_dy[0]);
   // printf("%f\n", output[pp3]);
}

__global__ void cuda_deriv42_z_thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int i;
   int j;

   if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } 
   if( (dev_sz[1]-3-3)<=z ){ return; } else { j = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp2 = IDX(i, j, dev_sz[2]-5); // IDX(i,j,ke-2)
   int pp3 = IDX(i, j, dev_sz[2]-6); // IDX(i,j,ke-3)
   int pp1 = IDX(i, j, dev_sz[2]-4); // IDX(i,j,ke-1)

   output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) * 0.50 / dev_dy[0];
   output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]+ 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];
   // printf("%f\n", output[pp1]);
}

__global__ void cuda_deriv42_xx_firstThreeForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } //i handler
    if( (dev_sz[1]-3-3)<=y ){ return; } else { j = y+3; } //j handler
    if( (dev_sz[2]-3-3)<=z ){ return; } else { k = z+1; } //k handler

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    int pp = IDX(i, j, k);

    output[pp] = (
        (-1)*dev_var_in[(*dev_u_offset) + pp - 2] 
        + 16.0*dev_var_in[(*dev_u_offset) + pp - 1] 
        - 30.0*dev_var_in[(*dev_u_offset) + pp] 
        + 16.0*dev_var_in[(*dev_u_offset) + pp + 1] 
        - dev_var_in[(*dev_u_offset) + pp + 2] 
    )*(1.0/(dev_dy[0]*dev_dy[0]))/12.0;

    // printf("%1.20f\n", output[pp]);
}

__global__ void cuda_deriv42_xx_secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int j;
   int k;

   if( (dev_sz[1]-3-3)<=x ){ return; } else { j = x+3; } 
   if( (dev_sz[2]-3-3)<=z ){ return; } else { k = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp3 = IDX(3, j, k); 
   int pp4 = IDX(4, j, k); 
   int pp5 = IDX(5, j, k); 
   int pp6 = IDX(6, j, k); 

   output[pp3] = (
            2.0 *   dev_var_in[(*dev_u_offset) + pp3] 
        -   5.0 *   dev_var_in[(*dev_u_offset) + pp4] 
        +   4.0 *   dev_var_in[(*dev_u_offset) + pp5] 
        -           dev_var_in[(*dev_u_offset) + pp6]
       ) * 1.0/(dev_dy[0]*dev_dy[0]);

   output[pp4] = (
                    dev_var_in[(*dev_u_offset) + pp3]
        -   2.0 *   dev_var_in[(*dev_u_offset) + pp4]
        +           dev_var_in[(*dev_u_offset) + pp5]
    ) * 1.0/(dev_dy[0]*dev_dy[0]);

//    printf("%f\n", output[pp3]);
}

__global__ void cuda_deriv42_xx_thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int k;
   int j;

   if( (dev_sz[2]-3-3)<=x ){ return; } else { k = x+3; } 
   if( (dev_sz[1]-3-3)<=z ){ return; } else { j = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp1 = IDX(dev_sz[0] - 4, j, k); // IDX(ie-1,j,k)
   int pp2 = IDX(dev_sz[0] - 5, j, k); // IDX(ie-2,j,k)
   int pp3 = IDX(dev_sz[0] - 6, j, k); // IDX(ie-3,j,k)
   int pp4 = IDX(dev_sz[0] - 7, j, k); // IDX(ie-4,j,k)

   output[pp2] = (
                    dev_var_in[(*dev_u_offset) + pp3] 
        -   2.0 *   dev_var_in[(*dev_u_offset) + pp2] 
        +           dev_var_in[(*dev_u_offset) + pp1] 
        ) * 1.0/(dev_dy[0]*dev_dy[0]);


    output[pp1] = (
        -   1.0 *   dev_var_in[(*dev_u_offset) + pp4] 
        +   4.0 *   dev_var_in[(*dev_u_offset) + pp3] 
        -   5.0 *   dev_var_in[(*dev_u_offset) + pp2] 
        +   2.0 *   dev_var_in[(*dev_u_offset) + pp1]
        ) * 1.0/(dev_dy[0]*dev_dy[0]);
   // printf("%f\n", output[pp1]);
}


__global__ void cuda_deriv42_yy_firstThreeForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } 
    if( (dev_sz[1]-3-3)<=y ){ return; } else { j = y+3; } 
    if( (dev_sz[2]-3-3)<=z ){ return; } else { k = z+1; } 

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    int pp = IDX(i, j, k);

    output[pp] = (
        (-1)*dev_var_in[(*dev_u_offset) + pp - 2*nx] 
        + 16.0*dev_var_in[(*dev_u_offset) + pp - nx] 
        - 30.0*dev_var_in[(*dev_u_offset) + pp] 
        + 16.0*dev_var_in[(*dev_u_offset) + pp + nx] 
        - dev_var_in[(*dev_u_offset) + pp + 2*nx] 
    )*(1.0/(dev_dy[0]*dev_dy[0]))/12.0;

    // printf("%1.20f\n", output[pp]);
}

__global__ void cuda_deriv42_yy_secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int i;
   int k;

   if( (dev_sz[1]-3-3)<=x ){ return; } else { i = x+3; } 
   if( (dev_sz[2]-3-3)<=z ){ return; } else { k = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp3 = IDX(i, 3, k); 
   int pp4 = IDX(i, 4, k); 
   int pp5 = IDX(i, 5, k); 
   int pp6 = IDX(i, 6, k); 

   output[pp3] = (
            2.0 *   dev_var_in[(*dev_u_offset) + pp3] 
        -   5.0 *   dev_var_in[(*dev_u_offset) + pp4] 
        +   4.0 *   dev_var_in[(*dev_u_offset) + pp5] 
        -           dev_var_in[(*dev_u_offset) + pp6]
       ) * 1.0/(dev_dy[0]*dev_dy[0]);

   output[pp4] = (
                    dev_var_in[(*dev_u_offset) + pp3]
        -   2.0 *   dev_var_in[(*dev_u_offset) + pp4]
        +           dev_var_in[(*dev_u_offset) + pp5]
    ) * 1.0/(dev_dy[0]*dev_dy[0]);

   // printf("%f\n", output[pp3]);
}

__global__ void cuda_deriv42_yy_thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int i;
   int k;

   if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } 
   if( (dev_sz[2]-3-3)<=z ){ return; } else { k = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp1 = IDX(i, dev_sz[1] - 4, k); 
   int pp2 = IDX(i, dev_sz[1] - 5, k); 
   int pp3 = IDX(i, dev_sz[1] - 6, k); 
   int pp4 = IDX(i, dev_sz[1] - 7, k); 

   output[pp2] = (
                    dev_var_in[(*dev_u_offset) + pp3] 
        -   2.0 *   dev_var_in[(*dev_u_offset) + pp2] 
        +           dev_var_in[(*dev_u_offset) + pp1] 
        ) * 1.0/(dev_dy[0]*dev_dy[0]);


    output[pp1] = (
        -   1.0 *   dev_var_in[(*dev_u_offset) + pp4] 
        +   4.0 *   dev_var_in[(*dev_u_offset) + pp3] 
        -   5.0 *   dev_var_in[(*dev_u_offset) + pp2] 
        +   2.0 *   dev_var_in[(*dev_u_offset) + pp1]
        ) * 1.0/(dev_dy[0]*dev_dy[0]);

   // printf("%f\n", output[pp1]);
}


__global__ void cuda_deriv42_zz_firstThreeForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
 {
    int x = threadIdx.x + blockIdx.x*10;
    int y = threadIdx.y + blockIdx.x*10;
    int z = threadIdx.z + blockIdx.x*10;

    int i;
    int j;
    int k;

    if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } 
    if( (dev_sz[1]-3-3)<=y ){ return; } else { j = y+3; } 
    if( (dev_sz[2]-3-3)<=z ){ return; } else { k = z+1; } 

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    int pp = IDX(i, j, k);

    int n = nx * ny;

    output[pp] = (
        (-1)*dev_var_in[(*dev_u_offset) + pp - 2*n] 
        + 16.0*dev_var_in[(*dev_u_offset) + pp - n] 
        - 30.0*dev_var_in[(*dev_u_offset) + pp] 
        + 16.0*dev_var_in[(*dev_u_offset) + pp + n] 
        - dev_var_in[(*dev_u_offset) + pp + 2*n] 
    )*(1.0/(dev_dy[0]*dev_dy[0]))/12.0;

    // printf("%1.20f\n", output[pp]);
}

__global__ void cuda_deriv42_zz_secondTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int j;
   int i;

   if( (dev_sz[1]-3-3)<=x ){ return; } else { j = x+3; } 
   if( (dev_sz[0]-3-3)<=z ){ return; } else { i = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp3 = IDX(i, j, 3); 
   int pp4 = IDX(i, j, 4); 
   int pp5 = IDX(i, j, 5); 
   int pp6 = IDX(i, j, 6); 

   output[pp3] = (
            2.0 *   dev_var_in[(*dev_u_offset) + pp3] 
        -   5.0 *   dev_var_in[(*dev_u_offset) + pp4] 
        +   4.0 *   dev_var_in[(*dev_u_offset) + pp5] 
        -           dev_var_in[(*dev_u_offset) + pp6]
       ) * 1.0/(dev_dy[0]*dev_dy[0]);

   output[pp4] = (
                    dev_var_in[(*dev_u_offset) + pp3]
        -   2.0 *   dev_var_in[(*dev_u_offset) + pp4]
        +           dev_var_in[(*dev_u_offset) + pp5]
    ) * 1.0/(dev_dy[0]*dev_dy[0]);

   // printf("%f\n", output[pp3]);
}

__global__ void cuda_deriv42_zz_thirdTwoForLoops(double * output, double * dev_var_in, const int * dev_u_offset, double * dev_dy, int * dev_sz)
{
   int x = threadIdx.x + blockIdx.x*30;
   int z = threadIdx.y + blockIdx.x*30;

   int i;
   int j;

   if( (dev_sz[0]-3-3)<=x ){ return; } else { i = x+3; } 
   if( (dev_sz[1]-3-3)<=z ){ return; } else { j = z+3; }

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   int pp1 = IDX(i, j, dev_sz[2] - 4); 
   int pp2 = IDX(i, j, dev_sz[2] - 5); 
   int pp3 = IDX(i, j, dev_sz[2] - 6); 
   int pp4 = IDX(i, j, dev_sz[2] - 7); 

   output[pp2] = (
                    dev_var_in[(*dev_u_offset) + pp3] 
        -   2.0 *   dev_var_in[(*dev_u_offset) + pp2] 
        +           dev_var_in[(*dev_u_offset) + pp1] 
        ) * 1.0/(dev_dy[0]*dev_dy[0]);


    output[pp1] = (
        -   1.0 *   dev_var_in[(*dev_u_offset) + pp4] 
        +   4.0 *   dev_var_in[(*dev_u_offset) + pp3] 
        -   5.0 *   dev_var_in[(*dev_u_offset) + pp2] 
        +   2.0 *   dev_var_in[(*dev_u_offset) + pp1]
        ) * 1.0/(dev_dy[0]*dev_dy[0]);
        
   // printf("%f\n", output[pp1]);
}


void cuda_deriv42_y(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
 {
    int zblocks = ((host_sz[2]-1)/10)+1;
    int yblocks = ((host_sz[0]-3)/10)+1;
    int xblocks = ((host_sz[1]-3)/10)+1;
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    cuda_deriv42_y_firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_y_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_y_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }


    if (bflag & (1u<<OCT_DIR_DOWN)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_y_secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_y_secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_y_secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }
    
    if (bflag & (1u<<OCT_DIR_UP)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_y_thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_y_thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_y_thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    //   for (int k = 3; k < sz[2]-3; k++) {
    //     for (int j = 3; j < sz[1]-3; j++) {
    //       for (int i = 3; i < sz[0]-3; i++) {
    //         int pp = IDX(i,j,k);
    //         if(std::isnan(Dyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //       }
    //     }
    //   }
    // #endif
 }
 
void cuda_deriv42_x(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
{
    int zblocks = ((host_sz[2]-1)/10)+1; // k
    int yblocks = ((host_sz[1]-1)/10)+1; // j
    int xblocks = ((host_sz[0]-3)/10)+1; // i
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    cuda_deriv42_x_firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_x_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_x_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    if (bflag & (1u<<OCT_DIR_LEFT)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[1]-1)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_x_secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_x_secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_x_secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }

    if (bflag & (1u<<OCT_DIR_RIGHT)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-1)/30)+1;
        int xblocks = ((host_sz[1]-1)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_x_thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_x_thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_x_thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    //     #pragma message("DEBUG_DERIVS_COMP: ON")
    //     for (int k = 3; k < sz[2]-3; k++) {
    //         for (int j = 3; j < sz[1]-3; j++) {
    //             for (int i = 3; i < sz[0]-3; i++) {
    //                 int pp = IDX(i,j,k);
    //                 if(isnan(Dxu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //                 }
    //             }
    //         }
    // #endif
}

void cuda_deriv42_z(double * output, double * dev_var_in, int * dev_u_offset, 
    double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
{
    int zblocks = ((host_sz[2]-1)/10)+1; // k
    int yblocks = ((host_sz[1]-1)/10)+1; // j
    int xblocks = ((host_sz[0]-3)/10)+1; // i
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    cuda_deriv42_z_firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_z_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_z_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    if (bflag & (1u<<OCT_DIR_BACK)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[0]-3)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_z_secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_z_secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_z_secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }

    if (bflag & (1u<<OCT_DIR_FRONT)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[0]-3)/30)+1;
        int xblocks = ((host_sz[0]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_z_thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_z_thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_z_thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    //   #ifdef DEBUG_DERIVS_COMP
    //     for (int k = kb; k < ke; k++) {
    //       for (int j = jb; j < je; j++) {
    //         for (int i = ib; i < ie; i++) {
    //           int pp = IDX(i,j,k);
    //           if(std::isnan(Dzu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //         }
    //       }
    //     }
    //   #endif
}

__global__ void calc_deriv42_adv_x(double * output, double * dev_var_in, int * dev_betax,
     double *dev_dx, int* dev_bflag, int* dev_sz, int* dev_u_offset) {
    
    //ib, jb, kb values are accumulated to the x, y, z
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 3 + threadIdx.y + blockIdx.x * blockDim.y;
    int k = 3 + threadIdx.z + blockIdx.x * blockDim.z;

    int idx_by_2 = 0.50 * (1.0 / dev_dx[0]);
    int idx_by_12 = (1.0 / dev_dx[0])/12;
    int nx = dev_sz[0];
    int ny = dev_sz[1];

    if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;
    
    int pp = IDX(i, j, k);
    //printf("pp = %f\n", dev_var_in[*dev_betax + pp]);
    
    if (dev_var_in[*dev_betax + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[*dev_u_offset + pp - 1]
                    - 10.0 * dev_var_in[*dev_u_offset + pp]
                    + 18.0 * dev_var_in[*dev_u_offset + pp + 1]
                    -  6.0 * dev_var_in[*dev_u_offset + pp + 2]
                    +        dev_var_in[*dev_u_offset + pp + 3]
                  ) * idx_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[*dev_u_offset + pp - 3]
                    +  6.0 * dev_var_in[*dev_u_offset + pp - 2]
                    - 18.0 * dev_var_in[*dev_u_offset + pp - 1]
                    + 10.0 * dev_var_in[*dev_u_offset + pp]
                    +  3.0 * dev_var_in[*dev_u_offset + pp +1]
                  ) * idx_by_12;
    }
    
    if ((*dev_bflag & (1u<<OCT_DIR_LEFT)) && (i == 0)) {
        
        output[IDX(3,j,k)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(3,j,k)]
                +  4.0 * dev_var_in[*dev_u_offset + IDX(4,j,k)]
                -        dev_var_in[*dev_u_offset + IDX(5,j,k)]
                ) * idx_by_2;

        if (dev_var_in[*dev_betax + IDX(4,j,k)] > 0.0) {
            output[IDX(4,j,k)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(4,j,k)]
                            +  4.0 * dev_var_in[*dev_u_offset + IDX(5,j,k)]
                            -        dev_var_in[*dev_u_offset + IDX(6,j,k)]
                        ) * idx_by_2;
        }
        else {
            output[IDX(4,j,k)] = ( -         dev_var_in[*dev_u_offset + IDX(3,j,k)]
                            +        dev_var_in[*dev_u_offset + IDX(5,j,k)]
                        ) * idx_by_2;
        }

        if (dev_var_in[*dev_betax + IDX(5,j,k)] > 0.0 ) {
            output[IDX(5,j,k)] = (-  3.0 * dev_var_in[*dev_u_offset + IDX(4,j,k)]
                        - 10.0 * dev_var_in[*dev_u_offset + IDX(5,j,k)]
                        + 18.0 * dev_var_in[*dev_u_offset + IDX(6,j,k)]
                        -  6.0 * dev_var_in[*dev_u_offset + IDX(7,j,k)]
                        +        dev_var_in[*dev_u_offset + IDX(8,j,k)]
                        ) * idx_by_12;
        }
        else {
            output[IDX(5,j,k)] = (           dev_var_in[*dev_u_offset + IDX(3,j,k)]
                            -  4.0 * dev_var_in[*dev_u_offset + IDX(4,j,k)]
                            +  3.0 * dev_var_in[*dev_u_offset + IDX(5,j,k)]
                        ) * idx_by_2;
        }
    }

    if ((*dev_bflag & (1u<<OCT_DIR_RIGHT)) && (i == 1)) {
        
        const int ie = nx - 3;
        
        if ( dev_var_in[*dev_betax + IDX(ie-3,j,k)] < 0.0 ) {
            output[IDX(ie-3,j,k)] = (  - 3.0 * dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                                    + 4.0 * dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                                    -       dev_var_in[*dev_u_offset + IDX(ie-1,j,k)]
                                 ) * idx_by_2;
        }
        else {
            output[IDX(ie-3,j,k)] = ( -   dev_var_in[*dev_u_offset + IDX(ie-6,j,k)]
                              +  6.0 * dev_var_in[*dev_u_offset + IDX(ie-5,j,k)]
                              - 18.0 * dev_var_in[*dev_u_offset + IDX(ie-4,j,k)]
                              + 10.0 * dev_var_in[*dev_u_offset + IDX(ie-3  ,j,k)]
                              +  3.0 * dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                            ) * idx_by_12;
        }
  
          if (dev_var_in[*dev_betax + IDX(ie-2,j,k)] > 0.0 ) {
            output[IDX(ie-2,j,k)] = (  -  dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                                    +  dev_var_in[*dev_u_offset + IDX(ie-1,j,k)]
                                 ) * idx_by_2;
          }
          else {
            output[IDX(ie-2,j,k)] = (     dev_var_in[*dev_u_offset + IDX(ie-4,j,k)]
                               - 4.0 * dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                               + 3.0 * dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                                 ) * idx_by_2;
          }
  
          output[IDX(ie-1,j,k)] = (          dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                                  - 4.0 * dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                                  + 3.0 * dev_var_in[*dev_u_offset + IDX(ie-1,j,k)]
                               ) * idx_by_2;
    }
}

void cuda_deriv42_adv_x(double * output, double * dev_var_in, 
    int * dev_u_offset, double * dev_dx, int * dev_sz,
    int * dev_betax, int* dev_bflag, const unsigned int * host_sz)
{
    hipError_t cudaStatus;
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction
    //printf("i = %d, j = %d, k = %d\n", ie, je, ke);

    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
    
    int requiredBlocks = maximumIterations / 10;
    if (ie % 10 != 0 || je % 10 != 0 || ke % 10 != 0) {
        requiredBlocks++;
    }
    
    int threads_x = ie / requiredBlocks;
    int threads_y = je / requiredBlocks;
    int threads_z = ke / requiredBlocks;
   
    calc_deriv42_adv_x <<< requiredBlocks, dim3(threads_x,threads_y,threads_z) >>> (output, dev_var_in, dev_betax,
        dev_dx, dev_bflag, dev_sz, dev_u_offset);
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_adv_x kernal!\n", cudaStatus);
            return;
    }
                    
    // hipMemcpy(Dxu, dev_Dxu, sizeof(double)*sizeof(Dxu), hipMemcpyDeviceToHost);

}

__global__ void calc_deriv42_adv_y(double * output, double * dev_var_in, int * dev_betay,
    double *dev_dy, int* dev_bflag, int* dev_sz, int* dev_u_offset) {
   
   //ib, jb, kb values are accumulated to the x, y, z
   int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
   int j = 3 + threadIdx.y + blockIdx.x * blockDim.y;
   int k = 3 + threadIdx.z + blockIdx.x * blockDim.z;

   int idy_by_2 = 0.50 * (1.0 / dev_dy[0]);
   int idy_by_12 = (1.0 / dev_dy[0])/12.0;
   int nx = dev_sz[0];
   int ny = dev_sz[1];
   
   if(i >= nx-3 || j >= dev_sz[1]-3 || k >= dev_sz[2]-3) return;
  
   int pp = IDX(i, j, k);
   //printf("pp = %f\n", dev_var_in[*dev_betax + pp]);
   if (dev_var_in[*dev_betay + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[*dev_u_offset + pp - nx]
                    - 10.0 * dev_var_in[*dev_u_offset + pp]
                    + 18.0 * dev_var_in[*dev_u_offset + pp + nx]
                    -  6.0 * dev_var_in[*dev_u_offset + pp + 2*nx]
                    +        dev_var_in[*dev_u_offset + pp + 3*nx]
                  ) * idy_by_12;
   }
   else {
       output[pp] = ( -        dev_var_in[*dev_u_offset + pp - 3*nx]
                   +  6.0 * dev_var_in[*dev_u_offset + pp - 2*nx]
                   - 18.0 * dev_var_in[*dev_u_offset + pp - nx]
                   + 10.0 * dev_var_in[*dev_u_offset + pp]
                   +  3.0 * dev_var_in[*dev_u_offset + pp +nx]
                 ) * idy_by_12;
               
   }
   
   if ((*dev_bflag & (1u<<OCT_DIR_DOWN)) && (j == 0)) {
       
       output[IDX(i,3,k)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(i,3,k)]
               +  4.0 * dev_var_in[*dev_u_offset + IDX(i,4,k)]
               -        dev_var_in[*dev_u_offset + IDX(i,5,k)]
               ) * idy_by_2;
               
       if (dev_var_in[*dev_betay + IDX(i,4,k)] > 0.0) {
           output[IDX(i,4,k)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(i,4,k)]
                           +  4.0 * dev_var_in[*dev_u_offset + IDX(i,5,k)]
                           -        dev_var_in[*dev_u_offset + IDX(i,6,k)]
                       ) * idy_by_2;

       }
       else {
           output[IDX(i,4,k)] = ( -         dev_var_in[*dev_u_offset + IDX(i,3,k)]
                           +        dev_var_in[*dev_u_offset + IDX(i,5,k)]
                       ) * idy_by_2;
                       
       }

       if (dev_var_in[*dev_betay + IDX(i,5,k)] > 0.0 ) {
           output[IDX(i,5,k)] = (-  3.0 * dev_var_in[*dev_u_offset + IDX(i,4,k)]
                       - 10.0 * dev_var_in[*dev_u_offset + IDX(i,5,k)]
                       + 18.0 * dev_var_in[*dev_u_offset + IDX(i,6,k)]
                       -  6.0 * dev_var_in[*dev_u_offset + IDX(i,7,k)]
                       +        dev_var_in[*dev_u_offset + IDX(i,8,k)]
                       ) * idy_by_12;
       }
       else {
           output[IDX(i,5,k)] = (           dev_var_in[*dev_u_offset + IDX(i,3,k)]
                           -  4.0 * dev_var_in[*dev_u_offset + IDX(i,4,k)]
                           +  3.0 * dev_var_in[*dev_u_offset + IDX(i,5,k)]
                       ) * idy_by_2;
       }
   }

   if ((*dev_bflag & (1u<<OCT_DIR_UP)) && (j == 1)) {
       
       const int je = dev_sz[1] - 3;
       
       if ( dev_var_in[*dev_betay + IDX(i,je-3,k)] < 0.0 ) {
           output[IDX(i,je-3,k)] = (  - 3.0 * dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                                   + 4.0 * dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                                   -       dev_var_in[*dev_u_offset + IDX(i,je-1,k)]
                                ) * idy_by_2;
       }
       else {
           output[IDX(i,je-3,k)] = ( -   dev_var_in[*dev_u_offset + IDX(i,je-6,k)]
                             +  6.0 * dev_var_in[*dev_u_offset + IDX(i,je-5,k)]
                             - 18.0 * dev_var_in[*dev_u_offset + IDX(i,je-4,k)]
                             + 10.0 * dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                             +  3.0 * dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                           ) * idy_by_12;
       }
 
         if (dev_var_in[*dev_betay + IDX(i,je-2,k)] > 0.0 ) {
           output[IDX(i,je-2,k)] = (  -  dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                                   +  dev_var_in[*dev_u_offset + IDX(i,je-1,k)]
                                ) * idy_by_2;
         }
         else {
           output[IDX(i,je-2,k)] = (     dev_var_in[*dev_u_offset + IDX(i,je-4,k)]
                              - 4.0 * dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                              + 3.0 * dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                                ) * idy_by_2;
         }
 
         output[IDX(i,je-1,k)]  = (          dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                                 - 4.0 * dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                                 + 3.0 * dev_var_in[*dev_u_offset + IDX(i,je-1,k)]
                              ) * idy_by_2;
   }
}

void cuda_deriv42_adv_y(double * output, double * dev_var_in, 
    int * dev_u_offset, double * dev_dy, int * dev_sz,
    int * dev_betay, int* dev_bflag, const unsigned int * host_sz)
{
    hipError_t cudaStatus;
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction
    //printf("i = %d, j = %d, k = %d\n", ie, je, ke);

    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
    
    int requiredBlocks = maximumIterations / 10;
    if (ie % 10 != 0 || je % 10 != 0 || ke % 10 != 0) {
        requiredBlocks++;
    }
    
    int threads_x = ie / requiredBlocks;
    int threads_y = je / requiredBlocks;
    int threads_z = ke / requiredBlocks;
    
    calc_deriv42_adv_y <<< requiredBlocks, dim3(threads_x,threads_y,threads_z) >>> (output, dev_var_in, dev_betay,
        dev_dy, dev_bflag, dev_sz, dev_u_offset);
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_adv_y kernal!\n", cudaStatus);
            return;
    }
                    
    // hipMemcpy(Dxu, dev_Dxu, sizeof(double)*sizeof(Dxu), hipMemcpyDeviceToHost);

}

__global__ void calc_deriv42_adv_z(double * output, double * dev_var_in, int * dev_betaz,
    double *dev_dz, int* dev_bflag, int* dev_sz, int* dev_u_offset) {
   
   //ib, jb, kb values are accumulated to the x, y, z
   int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
   int j = 3 + threadIdx.y + blockIdx.x * blockDim.y;
   int k = 3 + threadIdx.z + blockIdx.x * blockDim.z;

   int idz_by_2 = 0.50 * (1.0 / dev_dz[0]);
   int idz_by_12 = (1.0 / dev_dz[0])/12.0;
   int nx = dev_sz[0];
   int ny = dev_sz[1];
   
   if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

   int n = nx * ny;
   int pp = IDX(i, j, k);
   
   if (dev_var_in[*dev_betaz + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[*dev_u_offset + pp - n]
                    - 10.0 * dev_var_in[*dev_u_offset + pp]
                    + 18.0 * dev_var_in[*dev_u_offset + pp + n]
                    -  6.0 * dev_var_in[*dev_u_offset + pp + 2*n]
                    +        dev_var_in[*dev_u_offset + pp + 3*n]
                  ) * idz_by_12;
   }
   else {
       output[pp] = ( -        dev_var_in[*dev_u_offset + pp - 3*n]
                   +  6.0 * dev_var_in[*dev_u_offset + pp - 2*n]
                   - 18.0 * dev_var_in[*dev_u_offset + pp - n]
                   + 10.0 * dev_var_in[*dev_u_offset + pp]
                   +  3.0 * dev_var_in[*dev_u_offset + pp +n]
                 ) * idz_by_12;
               
   }
   
   if ((*dev_bflag & (1u<<OCT_DIR_BACK)) && (k == 0)) {
       
       output[IDX(i,j,3)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,3)]
               +  4.0 * dev_var_in[*dev_u_offset + IDX(i,j,4)]
               -        dev_var_in[*dev_u_offset + IDX(i,j,5)]
               ) * idz_by_2;
               
       if (dev_var_in[*dev_betaz + IDX(i,j,4)] > 0.0) {
           output[IDX(i,j,4)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,4)]
                           +  4.0 * dev_var_in[*dev_u_offset + IDX(i,j,5)]
                           -        dev_var_in[*dev_u_offset + IDX(i,j,6)]
                       ) * idz_by_2;

       }
       else {
           output[IDX(i,j,4)] = ( -         dev_var_in[*dev_u_offset + IDX(i,j,3)]
                           +        dev_var_in[*dev_u_offset + IDX(i,j,5)]
                       ) * idz_by_2;
                       
       }

       if (dev_var_in[*dev_betaz + IDX(i,j,5)] > 0.0 ) {
           output[IDX(i,j,5)] = (-  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,4)]
                       - 10.0 * dev_var_in[*dev_u_offset + IDX(i,j,5)]
                       + 18.0 * dev_var_in[*dev_u_offset + IDX(i,j,6)]
                       -  6.0 * dev_var_in[*dev_u_offset + IDX(i,j,7)]
                       +        dev_var_in[*dev_u_offset + IDX(i,j,8)]
                       ) * idz_by_12;
       }
       else {
           output[IDX(i,j,5)] = (           dev_var_in[*dev_u_offset + IDX(i,j,3)]
                           -  4.0 * dev_var_in[*dev_u_offset + IDX(i,j,4)]
                           +  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,5)]
                       ) * idz_by_2;
       }
   }

   if ((*dev_bflag & (1u<<OCT_DIR_FRONT)) && (k == 1)) {
       
       const int ke = dev_sz[12] - 3;
       
       if ( dev_var_in[*dev_betaz + IDX(i,j,ke-3)] < 0.0 ) {
           output[IDX(i,j,ke-3)] = (  - 3.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                                   + 4.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                                   -       dev_var_in[*dev_u_offset + IDX(i,j,ke-1)]
                                ) * idz_by_2;
       }
       else {
           output[IDX(i,j,ke-3)] = ( -   dev_var_in[*dev_u_offset + IDX(i,j,ke-6)]
                             +  6.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-5)]
                             - 18.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-4)]
                             + 10.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                             +  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                           ) * idz_by_12;
       }
 
         if (dev_var_in[*dev_betaz + IDX(i,j,ke-2)] > 0.0 ) {
           output[IDX(i,j,ke-2)] = (  -  dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                                   +  dev_var_in[*dev_u_offset + IDX(i,j,ke-1)]
                                ) * idz_by_2;
         }
         else {
           output[IDX(i,j,ke-2)] = (     dev_var_in[*dev_u_offset + IDX(i,j,ke-4)]
                              - 4.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                              + 3.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                                ) * idz_by_2;
         }
 
         output[IDX(i,j,ke-1)]  = (          dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                                 - 4.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                                 + 3.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-1)]
                              ) * idz_by_2;
   }
}

void cuda_deriv42_adv_z(double * output, double * dev_var_in, 
    int * dev_u_offset, double * dev_dz, int * dev_sz,
    int * dev_betaz, int* dev_bflag, const unsigned int * host_sz)
{
    hipError_t cudaStatus;
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction
    //printf("i = %d, j = %d, k = %d\n", ie, je, ke);

    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
    
    int requiredBlocks = maximumIterations / 10;
    if (ie % 10 != 0 || je % 10 != 0 || ke % 10 != 0) {
        requiredBlocks++;
    }
    
    int threads_x = ie / requiredBlocks;
    int threads_y = je / requiredBlocks;
    int threads_z = ke / requiredBlocks;
    
    calc_deriv42_adv_y <<< requiredBlocks, dim3(threads_x,threads_y,threads_z) >>> (output, dev_var_in, dev_betaz,
        dev_dz, dev_bflag, dev_sz, dev_u_offset);
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_adv_z kernal!\n", cudaStatus);
            return;
    }
                    
    // hipMemcpy(Dxu, dev_Dxu, sizeof(double)*sizeof(Dxu), hipMemcpyDeviceToHost);
}
 
void cuda_deriv42_xx(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
{
    int zblocks = ((host_sz[2]-3)/10)+1; // k
    int yblocks = ((host_sz[1]-3)/10)+1; // j
    int xblocks = ((host_sz[0]-3)/10)+1; // i
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    cuda_deriv42_xx_firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_xx_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_xx_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    if (bflag & (1u<<OCT_DIR_LEFT)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-3)/30)+1;
        int xblocks = ((host_sz[1]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_xx_secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_xx_secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_xx_secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }

    if (bflag & (1u<<OCT_DIR_RIGHT)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[2]-3)/30)+1;
        int xblocks = ((host_sz[1]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_xx_thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_xx_thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_xx_thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    // for (int k = kb; k < ke; k++) {
    //   for (int j = jb; j < je; j++) {
    //     for (int i = ib; i < ie; i++) {
    //       int pp = IDX(i,j,k);
    //       if(std::isnan(DxDxu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //     }
    //   }
    // }
    // #endif    
}

void cuda_deriv42_yy(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
{
    int zblocks = ((host_sz[2]-3)/10)+1; // k
    int yblocks = ((host_sz[1]-3)/10)+1; // j
    int xblocks = ((host_sz[0]-3)/10)+1; // i
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    cuda_deriv42_yy_firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_yy_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_yy_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    if (bflag & (1u<<OCT_DIR_DOWN)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[0]-3)/30)+1;
        int xblocks = ((host_sz[2]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_yy_secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_yy_secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_yy_secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }

    if (bflag & (1u<<OCT_DIR_UP)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[0]-3)/30)+1;
        int xblocks = ((host_sz[2]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_yy_thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_yy_thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_yy_thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    // for (int k = kb; k < ke; k++) {
    // for (int j = jb; j < je; j++) {
    //     for (int i = ib; i < ie; i++) {
    //     int pp = IDX(i,j,k);
    //     if(std::isnan(DyDyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //     }
    // }
    // }
    // #endif 
}

void cuda_deriv42_zz(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, int * dev_sz, unsigned bflag, const unsigned int * host_sz)
{
    int zblocks = ((host_sz[2]-3)/10)+1; // k
    int yblocks = ((host_sz[1]-3)/10)+1; // j
    int xblocks = ((host_sz[0]-3)/10)+1; // i
    int max1 = ( zblocks < yblocks ) ? yblocks : zblocks;
    int max = ( ( max1 < xblocks ) ? xblocks : max1 );

    cuda_deriv42_zz_firstThreeForLoops<<< max, dim3(10, 10, 10) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_zz_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_zz_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    if (bflag & (1u<<OCT_DIR_BACK)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[0]-3)/30)+1;
        int xblocks = ((host_sz[2]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_zz_secondTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_zz_secondTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_zz_secondTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    }

    if (bflag & (1u<<OCT_DIR_FRONT)) {
        // Not tested yet-----------------------------------------------------------------------------
        int yblocks = ((host_sz[0]-3)/30)+1;
        int xblocks = ((host_sz[2]-3)/30)+1;
        int max = ( xblocks < yblocks ) ? yblocks : xblocks;

        cuda_deriv42_zz_thirdTwoForLoops<<< max, dim3(30, 30) >>>(output, dev_var_in, dev_u_offset, dev_dy, dev_sz);

        // Check for any errors launching the kernel
        hipError_t cudaStatus;
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda_deriv42_zz_thirdTwoForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_zz_thirdTwoForLoops kernal!\n", cudaStatus);
            return;
        }
    } 

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    // for (int k = kb; k < ke; k++) {
    //   for (int j = jb; j < je; j++) {
    //     for (int i = ib; i < ie; i++) {
    //       int pp = IDX(i,j,k);
    //       if(std::isnan(DzDzu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //     }
    //   }
    // }
    // #endif
}