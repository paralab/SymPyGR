#include "hip/hip_runtime.h"
/**
 * Created on: March 15, 2018
 * 		Author: Akila
 **/

 #include "derivs_cuda.h"
 

 __global__ void calc_deriv42_x(double * output, double * dev_var_in, 
        const int * dev_u_offset, double * dev_dy, int * dev_sz, int* dev_bflag)
 {

    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 1 + threadIdx.y + blockIdx.y * blockDim.y;
    int k = 1 + threadIdx.z + blockIdx.z * blockDim.z;

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    if(i >= nx-3 || j >= ny-1 || k >= dev_sz[2]-1) return;
    
    int pp = IDX(i, j, k);
 
    output[pp] = (dev_var_in[(*dev_u_offset) + pp - 2] - 8.0*dev_var_in[(*dev_u_offset)
                     + pp - 1] + 8.0*dev_var_in[(*dev_u_offset) + pp + 1] 
                     - dev_var_in[(*dev_u_offset) + pp + 2] )*((1.0/dev_dy[0])/12.0);

    if ((*dev_bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
        int pp3 = IDX(3, j, k);
        int pp4 = IDX(4, j, k);
        int pp5 = IDX(5, j, k);
        output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] + 4*dev_var_in[(*dev_u_offset) 
                    + pp4] - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
        output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) 
                    + pp3]) * (0.50/dev_dy[0]);
    }

    if ((*dev_bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
        int pp2 = IDX(nx-5, j, k); // IDX(ie-2,j,k)
        int pp3 = IDX(nx-6, j, k); // IDX(ie-3,j,k)
        int pp1 = IDX(nx-4,j,k); // IDX(ie-1,j,k)
        output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) 
                    * 0.50 / dev_dy[0];
        output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]
                    + 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];

    }
    
 }

 void cuda_deriv42_x(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, 
        int * dev_sz, int* dev_bflag, const unsigned int * host_sz)
 {
 
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 1;//y direction
    const int ke = host_sz[2] - 1;//z direction
  
    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
     
    int requiredBlocks = (9+maximumIterations) / 10;
    
    calc_deriv42_x <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                      dim3((ie + requiredBlocks -1)/requiredBlocks,
                      (je + requiredBlocks -1)/requiredBlocks, 
                      (ke + requiredBlocks -1)/requiredBlocks) >>> (output, dev_var_in,
                        dev_u_offset, dev_dy, dev_sz, dev_bflag);
 
     // Check for any errors launching the kernel
     hipError_t cudaStatus;
     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "cuda_deriv42_x_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
         return;
     }
     // hipDeviceSynchronize waits for the kernel to finish, and returns
     // any errors encountered during the launch.
     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_x_firstThreeForLoops kernal!\n", cudaStatus);
         return;
     }
 
     // No GPU code for the following part
     // #ifdef DEBUG_DERIVS_COMP
     //     #pragma message("DEBUG_DERIVS_COMP: ON")
     //     for (int k = 3; k < sz[2]-3; k++) {
     //         for (int j = 3; j < sz[1]-3; j++) {
     //             for (int i = 3; i < sz[0]-3; i++) {
     //                 int pp = IDX(i,j,k);
     //                 if(isnan(Dxu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
     //                 }
     //             }
     //         }
     // #endif
 }

__global__ void calc_deriv42_y(double* output, double * dev_var_in, const int * dev_u_offset,
                         double * dev_dy, int * dev_sz, int* dev_bflag)
{
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int k = 1 + threadIdx.z + blockIdx.z * blockDim.z;

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-1) return;
    
    int pp = IDX(i, j, k);

    output[pp] = (dev_var_in[*dev_u_offset + pp - 2*nx] 
                - 8.0*dev_var_in[*dev_u_offset + pp - nx] 
                + 8.0*dev_var_in[*dev_u_offset + pp + nx] 
                - dev_var_in[*dev_u_offset + pp + 2*nx] )*((1.0/dev_dy[0])/12.0);
    
            
    if ((*dev_bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
        int pp3 = IDX(i, 3, k);
        int pp4 = IDX(i, 4, k);
        int pp5 = IDX(i, 5, k);

        output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] +  4*dev_var_in[(*dev_u_offset) + pp4] 
                    - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
        output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) + pp3]) 
                    * (0.50/dev_dy[0]);
        
    }

    if ((*dev_bflag & (1u<<OCT_DIR_UP)) && j==4)  {
        int pp2 = IDX(i, ny-5, k); // IDX(i,je-2,k)
        int pp3 = IDX(i, ny-6, k); // IDX(i,je-3,k)
        int pp1 = IDX(i, ny-4, k); // IDX(i,je-1,k)
    
        output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) 
                    * 0.50 / dev_dy[0];
        output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]
                    + 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];
        
    }
}

void cuda_deriv42_y(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, 
                int * dev_sz, int* dev_bflag, const unsigned int * host_sz)
 {
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 1;//z direction
  
    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
     
    int requiredBlocks = (9+maximumIterations) / 10;

    calc_deriv42_y<<<dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                    dim3((ie + requiredBlocks -1)/requiredBlocks,
                    (je + requiredBlocks -1)/requiredBlocks, 
                    (ke + requiredBlocks -1)/requiredBlocks)>>> 
                    (output, dev_var_in, dev_u_offset, dev_dy, dev_sz, dev_bflag);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "calc_deriv42_y Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calc_deriv42_y kernal!\n", cudaStatus);
        return;
    }

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    //   for (int k = 3; k < sz[2]-3; k++) {
    //     for (int j = 3; j < sz[1]-3; j++) {
    //       for (int i = 3; i < sz[0]-3; i++) {
    //         int pp = IDX(i,j,k);
    //         if(std::isnan(Dyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //       }
    //     }
    //   }
    // #endif
 }

__global__ void calc_deriv42_z(double * output, double * dev_var_in, const int * dev_u_offset,
                             double * dev_dy, int * dev_sz, int* dev_bflag)
{
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

    int n = nx * ny;
    int pp = IDX(i, j, k);
   
   output[pp] = (dev_var_in[(*dev_u_offset) + pp - 2*n] - 8.0*dev_var_in[(*dev_u_offset) + pp - n] 
                + 8.0*dev_var_in[(*dev_u_offset) + pp + n] - dev_var_in[(*dev_u_offset) + pp + 2*n]) 
                * ((1.0/dev_dy[0])/12);
    
    if ((*dev_bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
        int pp3 = IDX(i, j, 3); // IDX(i, j, 3)
        int pp4 = IDX(i, j, 4); // IDX(i,j,4)
        int pp5 = IDX(i, j, 5); // IDX(i,j,5)

        output[pp3] = ((-3)*dev_var_in[(*dev_u_offset) + pp3] + 4*dev_var_in[(*dev_u_offset) + pp4] 
                    - dev_var_in[(*dev_u_offset) + pp5]) * 0.5 / dev_dy[0];
        output[pp4] = (dev_var_in[(*dev_u_offset) + pp5] - dev_var_in[(*dev_u_offset) + pp3])
                     * (0.50/dev_dy[0]);
    }
            
    if ((*dev_bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
        int pp2 = IDX(i, j, dev_sz[2]-5); // IDX(i,j,ke-2)
        int pp3 = IDX(i, j, dev_sz[2]-6); // IDX(i,j,ke-3)
        int pp1 = IDX(i, j, dev_sz[2]-4); // IDX(i,j,ke-1)

        output[pp2] = (dev_var_in[(*dev_u_offset) + pp1] - dev_var_in[(*dev_u_offset) + pp3]) 
                    * 0.50 / dev_dy[0];
        output[pp1] = (dev_var_in[(*dev_u_offset) + pp3]- 4.0 * dev_var_in[(*dev_u_offset) + pp2]
                    + 3.0 * dev_var_in[(*dev_u_offset) + pp1]) * 0.50 / dev_dy[0];
    }
  
}

void cuda_deriv42_z(double * output, double * dev_var_in, int * dev_u_offset, 
    double * dev_dy, int * dev_sz, int* dev_bflag, const unsigned int * host_sz)
{
     const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction
  
    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
     
    int requiredBlocks = (9+maximumIterations) / 10;

    calc_deriv42_z<<<dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                    dim3((ie + requiredBlocks -1)/requiredBlocks,
                    (je + requiredBlocks -1)/requiredBlocks, 
                    (ke + requiredBlocks -1)/requiredBlocks)>>> 
                    (output, dev_var_in, dev_u_offset, dev_dy, dev_sz, dev_bflag);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_z_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_z_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    //   #ifdef DEBUG_DERIVS_COMP
    //     for (int k = kb; k < ke; k++) {
    //       for (int j = jb; j < je; j++) {
    //         for (int i = ib; i < ie; i++) {
    //           int pp = IDX(i,j,k);
    //           if(std::isnan(Dzu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //         }
    //       }
    //     }
    //   #endif
}

__global__ void calc_deriv42_xx(double * output, double * dev_var_in, const int * dev_u_offset,
             double * dev_dy, int * dev_sz, int* dev_bflag)
 {
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

    int pp = IDX(i, j, k);

    output[pp] = ((-1)*dev_var_in[(*dev_u_offset) + pp - 2] 
                + 16.0*dev_var_in[(*dev_u_offset) + pp - 1] 
                - 30.0*dev_var_in[(*dev_u_offset) + pp] 
                + 16.0*dev_var_in[(*dev_u_offset) + pp + 1] 
                - dev_var_in[(*dev_u_offset) + pp + 2] 
            )*(1.0/(dev_dy[0]*dev_dy[0]))/12.0;
            
    if ((*dev_bflag & (1u<<OCT_DIR_LEFT)) && i==3)  {
        int pp3 = IDX(3, j, k); 
        int pp4 = IDX(4, j, k); 
        int pp5 = IDX(5, j, k); 
        int pp6 = IDX(6, j, k); 
     
        output[pp3] = (
                 2.0 *   dev_var_in[(*dev_u_offset) + pp3] 
             -   5.0 *   dev_var_in[(*dev_u_offset) + pp4] 
             +   4.0 *   dev_var_in[(*dev_u_offset) + pp5] 
             -           dev_var_in[(*dev_u_offset) + pp6]
            ) * 1.0/(dev_dy[0]*dev_dy[0]);
     
        output[pp4] = (
                         dev_var_in[(*dev_u_offset) + pp3]
             -   2.0 *   dev_var_in[(*dev_u_offset) + pp4]
             +           dev_var_in[(*dev_u_offset) + pp5]
         ) * 1.0/(dev_dy[0]*dev_dy[0]);
     
    }
                    
    if ((*dev_bflag & (1u<<OCT_DIR_RIGHT)) && i==4)  {
        int pp1 = IDX(dev_sz[0] - 4, j, k); // IDX(ie-1,j,k)
        int pp2 = IDX(dev_sz[0] - 5, j, k); // IDX(ie-2,j,k)
        int pp3 = IDX(dev_sz[0] - 6, j, k); // IDX(ie-3,j,k)
        int pp4 = IDX(dev_sz[0] - 7, j, k); // IDX(ie-4,j,k)

        output[pp2] = (
                            dev_var_in[(*dev_u_offset) + pp3] 
                -   2.0 *   dev_var_in[(*dev_u_offset) + pp2] 
                +           dev_var_in[(*dev_u_offset) + pp1] 
                ) * 1.0/(dev_dy[0]*dev_dy[0]);


            output[pp1] = (
                -   1.0 *   dev_var_in[(*dev_u_offset) + pp4] 
                +   4.0 *   dev_var_in[(*dev_u_offset) + pp3] 
                -   5.0 *   dev_var_in[(*dev_u_offset) + pp2] 
                +   2.0 *   dev_var_in[(*dev_u_offset) + pp1]
                ) * 1.0/(dev_dy[0]*dev_dy[0]);
    }
}

void cuda_deriv42_xx(double * output, double * dev_var_in, int * dev_u_offset, 
                double * dev_dy, int * dev_sz, int* dev_bflag, const unsigned int * host_sz)
{
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction
  
    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
     
    int requiredBlocks = (9+maximumIterations) / 10;

    calc_deriv42_xx<<<dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                    dim3((ie + requiredBlocks -1)/requiredBlocks,
                    (je + requiredBlocks -1)/requiredBlocks, 
                    (ke + requiredBlocks -1)/requiredBlocks)>>> 
                    (output, dev_var_in, dev_u_offset, dev_dy, dev_sz, dev_bflag);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_xx_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_xx_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    // for (int k = kb; k < ke; k++) {
    //   for (int j = jb; j < je; j++) {
    //     for (int i = ib; i < ie; i++) {
    //       int pp = IDX(i,j,k);
    //       if(std::isnan(DxDxu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //     }
    //   }
    // }
    // #endif    
}



__global__ void calc_deriv42_yy(double * output, double * dev_var_in, const int * dev_u_offset, 
                double * dev_dy, int * dev_sz, int* dev_bflag)
 {
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

    int pp = IDX(i, j, k);

    output[pp] = ((-1)*dev_var_in[(*dev_u_offset) + pp - 2*nx] 
                + 16.0*dev_var_in[(*dev_u_offset) + pp - nx] 
                - 30.0*dev_var_in[(*dev_u_offset) + pp] 
                + 16.0*dev_var_in[(*dev_u_offset) + pp + nx] 
                - dev_var_in[(*dev_u_offset) + pp + 2*nx] 
            )*(1.0/(dev_dy[0]*dev_dy[0]))/12.0;
            
    if ((*dev_bflag & (1u<<OCT_DIR_DOWN)) && j==3)  {
        int pp3 = IDX(i, 3, k); 
        int pp4 = IDX(i, 4, k); 
        int pp5 = IDX(i, 5, k); 
        int pp6 = IDX(i, 6, k); 
     
        output[pp3] = (
                 2.0 *   dev_var_in[(*dev_u_offset) + pp3] 
             -   5.0 *   dev_var_in[(*dev_u_offset) + pp4] 
             +   4.0 *   dev_var_in[(*dev_u_offset) + pp5] 
             -           dev_var_in[(*dev_u_offset) + pp6]
            ) * 1.0/(dev_dy[0]*dev_dy[0]);
     
        output[pp4] = (
                         dev_var_in[(*dev_u_offset) + pp3]
             -   2.0 *   dev_var_in[(*dev_u_offset) + pp4]
             +           dev_var_in[(*dev_u_offset) + pp5]
         ) * 1.0/(dev_dy[0]*dev_dy[0]);
    }
                            
    if ((*dev_bflag & (1u<<OCT_DIR_UP)) && j==4)  {
        int pp1 = IDX(i, dev_sz[1] - 4, k); 
        int pp2 = IDX(i, dev_sz[1] - 5, k); 
        int pp3 = IDX(i, dev_sz[1] - 6, k); 
        int pp4 = IDX(i, dev_sz[1] - 7, k); 
     
        output[pp2] = (
                         dev_var_in[(*dev_u_offset) + pp3] 
             -   2.0 *   dev_var_in[(*dev_u_offset) + pp2] 
             +           dev_var_in[(*dev_u_offset) + pp1] 
             ) * 1.0/(dev_dy[0]*dev_dy[0]);
     
     
         output[pp1] = (
             -   1.0 *   dev_var_in[(*dev_u_offset) + pp4] 
             +   4.0 *   dev_var_in[(*dev_u_offset) + pp3] 
             -   5.0 *   dev_var_in[(*dev_u_offset) + pp2] 
             +   2.0 *   dev_var_in[(*dev_u_offset) + pp1]
             ) * 1.0/(dev_dy[0]*dev_dy[0]);
    
    }
    
}

void cuda_deriv42_yy(double * output, double * dev_var_in, int * dev_u_offset, double * dev_dy, 
                int * dev_sz, int* dev_bflag, const unsigned int * host_sz)
{
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction
  
    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
     
    int requiredBlocks = (9+maximumIterations) / 10;

    calc_deriv42_yy<<<dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                    dim3((ie + requiredBlocks -1)/requiredBlocks,
                    (je + requiredBlocks -1)/requiredBlocks, 
                    (ke + requiredBlocks -1)/requiredBlocks)>>> 
                    (output, dev_var_in, dev_u_offset, dev_dy, dev_sz, dev_bflag);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_yy_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_yy_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    // for (int k = kb; k < ke; k++) {
    // for (int j = jb; j < je; j++) {
    //     for (int i = ib; i < ie; i++) {
    //     int pp = IDX(i,j,k);
    //     if(std::isnan(DyDyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //     }
    // }
    // }
    // #endif 
}


__global__ void calc_deriv42_zz(double * output, double * dev_var_in, const int * dev_u_offset,
                 double * dev_dy, int * dev_sz, int* dev_bflag)
 {
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

    int nx = dev_sz[0]; 
    int ny = dev_sz[1]; 
    if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

    int pp = IDX(i, j, k);
    int n = nx * ny;

    output[pp] = ((-1)*dev_var_in[(*dev_u_offset) + pp - 2*n] 
                + 16.0*dev_var_in[(*dev_u_offset) + pp - n] 
                - 30.0*dev_var_in[(*dev_u_offset) + pp] 
                + 16.0*dev_var_in[(*dev_u_offset) + pp + n] 
                - dev_var_in[(*dev_u_offset) + pp + 2*n] 
            )*(1.0/(dev_dy[0]*dev_dy[0]))/12.0;

    if ((*dev_bflag & (1u<<OCT_DIR_BACK)) && k==3)  {
        int pp3 = IDX(i, j, 3); 
        int pp4 = IDX(i, j, 4); 
        int pp5 = IDX(i, j, 5); 
        int pp6 = IDX(i, j, 6); 
     
        output[pp3] = (
                 2.0 *   dev_var_in[(*dev_u_offset) + pp3] 
             -   5.0 *   dev_var_in[(*dev_u_offset) + pp4] 
             +   4.0 *   dev_var_in[(*dev_u_offset) + pp5] 
             -           dev_var_in[(*dev_u_offset) + pp6]
            ) * 1.0/(dev_dy[0]*dev_dy[0]);
     
        output[pp4] = (
                         dev_var_in[(*dev_u_offset) + pp3]
             -   2.0 *   dev_var_in[(*dev_u_offset) + pp4]
             +           dev_var_in[(*dev_u_offset) + pp5]
         ) * 1.0/(dev_dy[0]*dev_dy[0]);
    }
                                    
    if ((*dev_bflag & (1u<<OCT_DIR_FRONT)) && k==4)  {
        int pp1 = IDX(i, j, dev_sz[2] - 4); 
        int pp2 = IDX(i, j, dev_sz[2] - 5); 
        int pp3 = IDX(i, j, dev_sz[2] - 6); 
        int pp4 = IDX(i, j, dev_sz[2] - 7); 

        output[pp2] = (
                            dev_var_in[(*dev_u_offset) + pp3] 
                -   2.0 *   dev_var_in[(*dev_u_offset) + pp2] 
                +           dev_var_in[(*dev_u_offset) + pp1] 
                ) * 1.0/(dev_dy[0]*dev_dy[0]);


            output[pp1] = (
                -   1.0 *   dev_var_in[(*dev_u_offset) + pp4] 
                +   4.0 *   dev_var_in[(*dev_u_offset) + pp3] 
                -   5.0 *   dev_var_in[(*dev_u_offset) + pp2] 
                +   2.0 *   dev_var_in[(*dev_u_offset) + pp1]
                ) * 1.0/(dev_dy[0]*dev_dy[0]);
    }
}


void cuda_deriv42_zz(double * output, double * dev_var_in, int * dev_u_offset, 
        double * dev_dy, int * dev_sz, int* dev_bflag, const unsigned int * host_sz)
{
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction
  
    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
     
    int requiredBlocks = (9+maximumIterations) / 10;

    calc_deriv42_zz<<<dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                    dim3((ie + requiredBlocks -1)/requiredBlocks,
                    (je + requiredBlocks -1)/requiredBlocks, 
                    (ke + requiredBlocks -1)/requiredBlocks)>>> 
                    (output, dev_var_in, dev_u_offset, dev_dy, dev_sz, dev_bflag);

    // Check for any errors launching the kernel
    hipError_t cudaStatus;
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cuda_deriv42_zz_firstThreeForLoops Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_zz_firstThreeForLoops kernal!\n", cudaStatus);
        return;
    }

    // No GPU code for the following part
    // #ifdef DEBUG_DERIVS_COMP
    // for (int k = kb; k < ke; k++) {
    //   for (int j = jb; j < je; j++) {
    //     for (int i = ib; i < ie; i++) {
    //       int pp = IDX(i,j,k);
    //       if(std::isnan(DzDzu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
    //     }
    //   }
    // }
    // #endif
}


__global__ void calc_deriv42_adv_x(double * output, double * dev_var_in, int * dev_betax,
     double *dev_dx, int* dev_bflag, int* dev_sz, int* dev_u_offset) {
    
    //ib, jb, kb values are accumulated to the x, y, z
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
    int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

    double idx_by_2 = 0.50 * (1.0 / dev_dx[0]);
    double idx_by_12 = (1.0 / dev_dx[0])/12;
    int nx = dev_sz[0];
    int ny = dev_sz[1];

    if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;
    
    int pp = IDX(i, j, k);
    //printf("ie = %d, je = %d, ke = %d\n", i, j, k);
    if (dev_var_in[*dev_betax + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[*dev_u_offset + pp - 1]
                    - 10.0 * dev_var_in[*dev_u_offset + pp]
                    + 18.0 * dev_var_in[*dev_u_offset + pp + 1]
                    -  6.0 * dev_var_in[*dev_u_offset + pp + 2]
                    +        dev_var_in[*dev_u_offset + pp + 3]
                  ) * idx_by_12;
    }
    else {
        output[pp] = ( -        dev_var_in[*dev_u_offset + pp - 3]
                    +  6.0 * dev_var_in[*dev_u_offset + pp - 2]
                    - 18.0 * dev_var_in[*dev_u_offset + pp - 1]
                    + 10.0 * dev_var_in[*dev_u_offset + pp]
                    +  3.0 * dev_var_in[*dev_u_offset + pp +1]
                  ) * idx_by_12;
    }
    
    if ((*dev_bflag & (1u<<OCT_DIR_LEFT)) && (i == 3)) {
        
        output[IDX(3,j,k)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(3,j,k)]
                +  4.0 * dev_var_in[*dev_u_offset + IDX(4,j,k)]
                -        dev_var_in[*dev_u_offset + IDX(5,j,k)]
                ) * idx_by_2;

        if (dev_var_in[*dev_betax + IDX(4,j,k)] > 0.0) {
            output[IDX(4,j,k)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(4,j,k)]
                            +  4.0 * dev_var_in[*dev_u_offset + IDX(5,j,k)]
                            -        dev_var_in[*dev_u_offset + IDX(6,j,k)]
                        ) * idx_by_2;
        }
        else {
            output[IDX(4,j,k)] = ( -         dev_var_in[*dev_u_offset + IDX(3,j,k)]
                            +        dev_var_in[*dev_u_offset + IDX(5,j,k)]
                        ) * idx_by_2;
        }

        if (dev_var_in[*dev_betax + IDX(5,j,k)] > 0.0 ) {
            output[IDX(5,j,k)] = (-  3.0 * dev_var_in[*dev_u_offset + IDX(4,j,k)]
                        - 10.0 * dev_var_in[*dev_u_offset + IDX(5,j,k)]
                        + 18.0 * dev_var_in[*dev_u_offset + IDX(6,j,k)]
                        -  6.0 * dev_var_in[*dev_u_offset + IDX(7,j,k)]
                        +        dev_var_in[*dev_u_offset + IDX(8,j,k)]
                        ) * idx_by_12;
        }
        else {
            output[IDX(5,j,k)] = (           dev_var_in[*dev_u_offset + IDX(3,j,k)]
                            -  4.0 * dev_var_in[*dev_u_offset + IDX(4,j,k)]
                            +  3.0 * dev_var_in[*dev_u_offset + IDX(5,j,k)]
                        ) * idx_by_2;
        }
    }

    if ((*dev_bflag & (1u<<OCT_DIR_RIGHT)) && (i == 4)) {
        
        const int ie = nx - 3;
        
        if ( dev_var_in[*dev_betax + IDX(ie-3,j,k)] < 0.0 ) {
            output[IDX(ie-3,j,k)] = (  - 3.0 * dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                                    + 4.0 * dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                                    -       dev_var_in[*dev_u_offset + IDX(ie-1,j,k)]
                                 ) * idx_by_2;
        }
        else {
            output[IDX(ie-3,j,k)] = ( -   dev_var_in[*dev_u_offset + IDX(ie-6,j,k)]
                              +  6.0 * dev_var_in[*dev_u_offset + IDX(ie-5,j,k)]
                              - 18.0 * dev_var_in[*dev_u_offset + IDX(ie-4,j,k)]
                              + 10.0 * dev_var_in[*dev_u_offset + IDX(ie-3  ,j,k)]
                              +  3.0 * dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                            ) * idx_by_12;
        }
  
          if (dev_var_in[*dev_betax + IDX(ie-2,j,k)] > 0.0 ) {
            output[IDX(ie-2,j,k)] = (  -  dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                                    +  dev_var_in[*dev_u_offset + IDX(ie-1,j,k)]
                                 ) * idx_by_2;
          }
          else {
            output[IDX(ie-2,j,k)] = (     dev_var_in[*dev_u_offset + IDX(ie-4,j,k)]
                               - 4.0 * dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                               + 3.0 * dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                                 ) * idx_by_2;
          }
  
          output[IDX(ie-1,j,k)] = (          dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                                  - 4.0 * dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                                  + 3.0 * dev_var_in[*dev_u_offset + IDX(ie-1,j,k)]
                               ) * idx_by_2;
    }
}

void cuda_deriv42_adv_x(double * output, double * dev_var_in, 
    int * dev_u_offset, double * dev_dx, int * dev_sz,
    int * dev_betax, int* dev_bflag, const unsigned int * host_sz)
{
    hipError_t cudaStatus;
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction
    //printf("ie = %d, je = %d, ke = %d\n", ie, je, ke);

    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
    
    int requiredBlocks = (9+maximumIterations) / 10;
  
    calc_deriv42_adv_x <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                        dim3((ie + requiredBlocks -1)/requiredBlocks,
                        (je + requiredBlocks -1)/requiredBlocks, 
                        (ke + requiredBlocks -1)/requiredBlocks) >>> (output, dev_var_in, dev_betax,
                            dev_dx, dev_bflag, dev_sz, dev_u_offset);
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_adv_x kernal!\n", cudaStatus);
            return;
    }
                    
    // hipMemcpy(Dxu, dev_Dxu, sizeof(double)*sizeof(Dxu), hipMemcpyDeviceToHost);

}

__global__ void calc_deriv42_adv_y(double * output, double * dev_var_in, int * dev_betay,
    double *dev_dy, int* dev_bflag, int* dev_sz, int* dev_u_offset) {
   
   //ib, jb, kb values are accumulated to the x, y, z
   int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
   int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
   int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

   double idy_by_2 = 0.50 * (1.0 / dev_dy[0]);
   double idy_by_12 = (1.0 / dev_dy[0])/12.0;
   int nx = dev_sz[0];
   int ny = dev_sz[1];
   
   if(i >= nx-3 || j >= dev_sz[1]-3 || k >= dev_sz[2]-3) return;
  
   int pp = IDX(i, j, k);
   //printf("pp = %f\n", dev_var_in[*dev_betax + pp]);
   if (dev_var_in[*dev_betay + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[*dev_u_offset + pp - nx]
                    - 10.0 * dev_var_in[*dev_u_offset + pp]
                    + 18.0 * dev_var_in[*dev_u_offset + pp + nx]
                    -  6.0 * dev_var_in[*dev_u_offset + pp + 2*nx]
                    +        dev_var_in[*dev_u_offset + pp + 3*nx]
                  ) * idy_by_12;
   }
   else {
       output[pp] = ( -        dev_var_in[*dev_u_offset + pp - 3*nx]
                   +  6.0 * dev_var_in[*dev_u_offset + pp - 2*nx]
                   - 18.0 * dev_var_in[*dev_u_offset + pp - nx]
                   + 10.0 * dev_var_in[*dev_u_offset + pp]
                   +  3.0 * dev_var_in[*dev_u_offset + pp +nx]
                 ) * idy_by_12;
               
   }
   
   if ((*dev_bflag & (1u<<OCT_DIR_DOWN)) && (j == 3)) {
       
       output[IDX(i,3,k)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(i,3,k)]
               +  4.0 * dev_var_in[*dev_u_offset + IDX(i,4,k)]
               -        dev_var_in[*dev_u_offset + IDX(i,5,k)]
               ) * idy_by_2;
               
       if (dev_var_in[*dev_betay + IDX(i,4,k)] > 0.0) {
           output[IDX(i,4,k)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(i,4,k)]
                           +  4.0 * dev_var_in[*dev_u_offset + IDX(i,5,k)]
                           -        dev_var_in[*dev_u_offset + IDX(i,6,k)]
                       ) * idy_by_2;

       }
       else {
           output[IDX(i,4,k)] = ( -         dev_var_in[*dev_u_offset + IDX(i,3,k)]
                           +        dev_var_in[*dev_u_offset + IDX(i,5,k)]
                       ) * idy_by_2;
                       
       }

       if (dev_var_in[*dev_betay + IDX(i,5,k)] > 0.0 ) {
           output[IDX(i,5,k)] = (-  3.0 * dev_var_in[*dev_u_offset + IDX(i,4,k)]
                       - 10.0 * dev_var_in[*dev_u_offset + IDX(i,5,k)]
                       + 18.0 * dev_var_in[*dev_u_offset + IDX(i,6,k)]
                       -  6.0 * dev_var_in[*dev_u_offset + IDX(i,7,k)]
                       +        dev_var_in[*dev_u_offset + IDX(i,8,k)]
                       ) * idy_by_12;
       }
       else {
           output[IDX(i,5,k)] = (           dev_var_in[*dev_u_offset + IDX(i,3,k)]
                           -  4.0 * dev_var_in[*dev_u_offset + IDX(i,4,k)]
                           +  3.0 * dev_var_in[*dev_u_offset + IDX(i,5,k)]
                       ) * idy_by_2;
       }
   }

   if ((*dev_bflag & (1u<<OCT_DIR_UP)) && (j == 4)) {
       
       const int je = dev_sz[1] - 3;
       
       if ( dev_var_in[*dev_betay + IDX(i,je-3,k)] < 0.0 ) {
           output[IDX(i,je-3,k)] = (  - 3.0 * dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                                   + 4.0 * dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                                   -       dev_var_in[*dev_u_offset + IDX(i,je-1,k)]
                                ) * idy_by_2;
       }
       else {
           output[IDX(i,je-3,k)] = ( -   dev_var_in[*dev_u_offset + IDX(i,je-6,k)]
                             +  6.0 * dev_var_in[*dev_u_offset + IDX(i,je-5,k)]
                             - 18.0 * dev_var_in[*dev_u_offset + IDX(i,je-4,k)]
                             + 10.0 * dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                             +  3.0 * dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                           ) * idy_by_12;
       }
 
         if (dev_var_in[*dev_betay + IDX(i,je-2,k)] > 0.0 ) {
           output[IDX(i,je-2,k)] = (  -  dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                                   +  dev_var_in[*dev_u_offset + IDX(i,je-1,k)]
                                ) * idy_by_2;
         }
         else {
           output[IDX(i,je-2,k)] = (     dev_var_in[*dev_u_offset + IDX(i,je-4,k)]
                              - 4.0 * dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                              + 3.0 * dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                                ) * idy_by_2;
         }
 
         output[IDX(i,je-1,k)]  = (          dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                                 - 4.0 * dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                                 + 3.0 * dev_var_in[*dev_u_offset + IDX(i,je-1,k)]
                              ) * idy_by_2;
   }
}

void cuda_deriv42_adv_y(double * output, double * dev_var_in, 
    int * dev_u_offset, double * dev_dy, int * dev_sz,
    int * dev_betay, int* dev_bflag, const unsigned int * host_sz)
{
    hipError_t cudaStatus;
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction

    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
    
    int requiredBlocks = (9+maximumIterations) / 10;
  
    calc_deriv42_adv_y <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                        dim3((ie + requiredBlocks -1)/requiredBlocks,
                        (je + requiredBlocks -1)/requiredBlocks, 
                        (ke + requiredBlocks -1)/requiredBlocks) >>> (output, dev_var_in, dev_betay,
                            dev_dy, dev_bflag, dev_sz, dev_u_offset);
        
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_adv_y kernal!\n", cudaStatus);
            return;
    }
                    
    // hipMemcpy(Dxu, dev_Dxu, sizeof(double)*sizeof(Dxu), hipMemcpyDeviceToHost);

}

__global__ void calc_deriv42_adv_z(double * output, double * dev_var_in, int * dev_betaz,
    double *dev_dz, int* dev_bflag, int* dev_sz, int* dev_u_offset) {
   
   //ib, jb, kb values are accumulated to the x, y, z
   int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
   int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
   int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

   double idz_by_2 = 0.50 * (1.0 / dev_dz[0]);
   double idz_by_12 = (1.0 / dev_dz[0])/12.0;
   int nx = dev_sz[0];
   int ny = dev_sz[1];
   
   if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

   int n = nx * ny;
   int pp = IDX(i, j, k);
   
   if (dev_var_in[*dev_betaz + pp] > 0.0 ) {
        output[pp] = ( -  3.0 * dev_var_in[*dev_u_offset + pp - n]
                    - 10.0 * dev_var_in[*dev_u_offset + pp]
                    + 18.0 * dev_var_in[*dev_u_offset + pp + n]
                    -  6.0 * dev_var_in[*dev_u_offset + pp + 2*n]
                    +        dev_var_in[*dev_u_offset + pp + 3*n]
                  ) * idz_by_12;
   }
   else {
       output[pp] = ( -        dev_var_in[*dev_u_offset + pp - 3*n]
                   +  6.0 * dev_var_in[*dev_u_offset + pp - 2*n]
                   - 18.0 * dev_var_in[*dev_u_offset + pp - n]
                   + 10.0 * dev_var_in[*dev_u_offset + pp]
                   +  3.0 * dev_var_in[*dev_u_offset + pp +n]
                 ) * idz_by_12;
               
   }
   
   if ((*dev_bflag & (1u<<OCT_DIR_BACK)) && (k == 3)) {
       
       output[IDX(i,j,3)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,3)]
               +  4.0 * dev_var_in[*dev_u_offset + IDX(i,j,4)]
               -        dev_var_in[*dev_u_offset + IDX(i,j,5)]
               ) * idz_by_2;
               
       if (dev_var_in[*dev_betaz + IDX(i,j,4)] > 0.0) {
           output[IDX(i,j,4)] = ( -  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,4)]
                           +  4.0 * dev_var_in[*dev_u_offset + IDX(i,j,5)]
                           -        dev_var_in[*dev_u_offset + IDX(i,j,6)]
                       ) * idz_by_2;

       }
       else {
           output[IDX(i,j,4)] = ( -         dev_var_in[*dev_u_offset + IDX(i,j,3)]
                           +        dev_var_in[*dev_u_offset + IDX(i,j,5)]
                       ) * idz_by_2;
                       
       }

       if (dev_var_in[*dev_betaz + IDX(i,j,5)] > 0.0 ) {
           output[IDX(i,j,5)] = (-  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,4)]
                       - 10.0 * dev_var_in[*dev_u_offset + IDX(i,j,5)]
                       + 18.0 * dev_var_in[*dev_u_offset + IDX(i,j,6)]
                       -  6.0 * dev_var_in[*dev_u_offset + IDX(i,j,7)]
                       +        dev_var_in[*dev_u_offset + IDX(i,j,8)]
                       ) * idz_by_12;
       }
       else {
           output[IDX(i,j,5)] = (           dev_var_in[*dev_u_offset + IDX(i,j,3)]
                           -  4.0 * dev_var_in[*dev_u_offset + IDX(i,j,4)]
                           +  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,5)]
                       ) * idz_by_2;
       }
   }

   if ((*dev_bflag & (1u<<OCT_DIR_FRONT)) && (k == 4)) {
       
       const int ke = dev_sz[12] - 3;
       
       if ( dev_var_in[*dev_betaz + IDX(i,j,ke-3)] < 0.0 ) {
           output[IDX(i,j,ke-3)] = (  - 3.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                                   + 4.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                                   -       dev_var_in[*dev_u_offset + IDX(i,j,ke-1)]
                                ) * idz_by_2;
       }
       else {
           output[IDX(i,j,ke-3)] = ( -   dev_var_in[*dev_u_offset + IDX(i,j,ke-6)]
                             +  6.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-5)]
                             - 18.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-4)]
                             + 10.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                             +  3.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                           ) * idz_by_12;
       }
 
         if (dev_var_in[*dev_betaz + IDX(i,j,ke-2)] > 0.0 ) {
           output[IDX(i,j,ke-2)] = (  -  dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                                   +  dev_var_in[*dev_u_offset + IDX(i,j,ke-1)]
                                ) * idz_by_2;
         }
         else {
           output[IDX(i,j,ke-2)] = (     dev_var_in[*dev_u_offset + IDX(i,j,ke-4)]
                              - 4.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                              + 3.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                                ) * idz_by_2;
         }
 
         output[IDX(i,j,ke-1)]  = (          dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                                 - 4.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                                 + 3.0 * dev_var_in[*dev_u_offset + IDX(i,j,ke-1)]
                              ) * idz_by_2;
   }
}

void cuda_deriv42_adv_z(double * output, double * dev_var_in, 
    int * dev_u_offset, double * dev_dz, int * dev_sz,
    int * dev_betaz, int* dev_bflag, const unsigned int * host_sz)
{
    hipError_t cudaStatus;
    const int ie = host_sz[0] - 3;//x direction
    const int je = host_sz[1] - 3;//y direction
    const int ke = host_sz[2] - 3;//z direction

    int temp_max = (ie>je)? ie : je;
    int maximumIterations = (temp_max>ke) ? temp_max: ke;
    
    int requiredBlocks = (9+maximumIterations) / 10;
  
    calc_deriv42_adv_z <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                        dim3((ie + requiredBlocks -1)/requiredBlocks,
                        (je + requiredBlocks -1)/requiredBlocks, 
                        (ke + requiredBlocks -1)/requiredBlocks) >>> (output, dev_var_in, dev_betaz,
                            dev_dz, dev_bflag, dev_sz, dev_u_offset);
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_deriv42_adv_z kernal!\n", cudaStatus);
            return;
    }
}

__global__ void calc_ko_deriv42_x(double * output, double * dev_var_in,
    double *dev_dx, int* dev_bflag, int* dev_sz, int* dev_u_offset) {
   
   //ib, jb, kb values are accumulated to the x, y, z
   int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
   int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
   int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

   int pp = IDX(i, j, k);
   
   output[pp] = (-1.0 / 64.0 / dev_dx[0]) *
                         (
                         -      dev_var_in[*dev_u_offset + pp - 3]
                         +  6.0*dev_var_in[*dev_u_offset + pp - 2]
                         - 15.0*dev_var_in[*dev_u_offset + pp - 1]
                         + 20.0*dev_var_in[*dev_u_offset + pp ]
                         - 15.0*dev_var_in[*dev_u_offset + pp + 1]
                         +  6.0*dev_var_in[*dev_u_offset + pp + 2]
                         -      dev_var_in[*dev_u_offset + pp + 3]
                         );
   
   if ((*dev_bflag & (1u<<OCT_DIR_LEFT)) && (i == 3)) {

    output[IDX(3,j,k)] =  (      dev_var_in[*dev_u_offset + IDX(6,j,k)]
                                - 3.0*dev_var_in[*dev_u_offset + IDX(5,j,k)]
                                + 3.0*dev_var_in[*dev_u_offset + IDX(4,j,k)]
                                -     dev_var_in[*dev_u_offset + IDX(3,j,k)]
                            )/59.0/48.0*64*dev_dx[0];
    output[IDX(4,j,k)] =  (     dev_var_in[*dev_u_offset + IDX(7,j,k)]
                                -  6.0*dev_var_in[*dev_u_offset + IDX(6,j,k)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(5,j,k)]
                                - 10.0*dev_var_in[*dev_u_offset + IDX(4,j,k)]
                                +  3.0*dev_var_in[*dev_u_offset + IDX(3,j,k)]
                                )/43.0/48.0*64*dev_dx[0];
    output[IDX(5,j,k)] =  (     dev_var_in[*dev_u_offset + IDX(8,j,k)]
                                -  6.0*dev_var_in[*dev_u_offset + IDX(7,j,k)]
                                + 15.0*dev_var_in[*dev_u_offset + IDX(6,j,k)]
                                - 19.0*dev_var_in[*dev_u_offset + IDX(5,j,k)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(4,j,k)]
                                -  3.0*dev_var_in[*dev_u_offset + IDX(3,j,k)]
                                )/49.0/48.0*64*dev_dx[0];
    }

   if ((*dev_bflag & (1u<<OCT_DIR_RIGHT)) && (i == 4)) {
       
       const int ie = nx - 3;
       output[IDX(ie-3,j,k)] = ( dev_var_in[*dev_u_offset + IDX(ie-6,j,k)]
                                - 6.0*dev_var_in[*dev_u_offset + IDX(ie-5,j,k)]
                                + 15.0*dev_var_in[*dev_u_offset + IDX(ie-4,j,k)]
                                - 19.0*dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                                -  3.0*dev_var_in[*dev_u_offset + IDX(ie-1,j,k)]
                                )/49.0/48.0*64*dev_dx[0];
        
        output[IDX(ie-2,j,k)] =  ( dev_var_in[*dev_u_offset + IDX(ie-5,j,k)]
                                -  6.0*dev_var_in[*dev_u_offset + IDX(ie-4,j,k)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                                - 10.0*dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                                +  3.0*dev_var_in[*dev_u_offset + IDX(ie-1,j,k)]
                                )/43.0/48.0*64*dev_dx[0];
       
 
        output[IDX(ie-1,j,k)] = ( dev_var_in[*dev_u_offset + IDX(ie-4,j,k)]
                                -  3.0*dev_var_in[*dev_u_offset + IDX(ie-3,j,k)]
                                +  3.0*dev_var_in[*dev_u_offset + IDX(ie-2,j,k)]
                                -      dev_var_in[*dev_u_offset + IDX(ie-1,j,k)]
                                )/59.0/48.0*64*dev_dx[0];
   }
}

void cuda_ko_deriv42_x(double * output, double * dev_var_in, 
   int * dev_u_offset, double * dev_dx, int * dev_sz,
   int* dev_bflag, const unsigned int * host_sz)
{
   hipError_t cudaStatus;
   const int ie = host_sz[0] - 3;//x direction
   const int je = host_sz[1] - 3;//y direction
   const int ke = host_sz[2] - 3;//z direction

   int temp_max = (ie>je)? ie : je;
   int maximumIterations = (temp_max>ke) ? temp_max: ke;

   int requiredBlocks = (9+maximumIterations) / 10;
  
   calc_ko_deriv42_x <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                        dim3((ie + requiredBlocks -1)/requiredBlocks,
                        (je + requiredBlocks -1)/requiredBlocks, 
                        (ke + requiredBlocks -1)/requiredBlocks) >>> (output, dev_var_in,
                            dev_dx, dev_bflag, dev_sz, dev_u_offset);
   
   cudaStatus = hipDeviceSynchronize();
   if (cudaStatus != hipSuccess) {
           fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_ko_deriv42_x kernal!\n", cudaStatus);
           return;
   }

}

__global__ void calc_ko_deriv42_y(double * output, double * dev_var_in,
    double *dev_dy, int* dev_bflag, int* dev_sz, int* dev_u_offset) {
   
   //ib, jb, kb values are accumulated to the x, y, z
   int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
   int j = 3 + threadIdx.y + blockIdx.x * blockDim.y;
   int k = 3 + threadIdx.z + blockIdx.x * blockDim.z;

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

   int pp = IDX(i, j, k);
   
   output[pp] = (-1.0 / 64.0 / dev_dy[0]) *
                (
                    -      dev_var_in[*dev_u_offset + pp-3*nx]
                    +  6.0*dev_var_in[*dev_u_offset + pp-2*nx]
                    - 15.0*dev_var_in[*dev_u_offset + pp-nx]
                    + 20.0*dev_var_in[*dev_u_offset + pp]
                    - 15.0*dev_var_in[*dev_u_offset + pp+nx]
                    +  6.0*dev_var_in[*dev_u_offset + pp+2*nx]
                    -      dev_var_in[*dev_u_offset + pp+3*nx]
                    );

   
   if ((*dev_bflag & (1u<<OCT_DIR_DOWN)) && (j == 3)) {

    output[IDX(i,3,k)] =  (      dev_var_in[*dev_u_offset +IDX(i,6,k)]
                                - 3.0*dev_var_in[*dev_u_offset +IDX(i,5,k)]
                                + 3.0*dev_var_in[*dev_u_offset + IDX(i,4,k)]
                                -     dev_var_in[*dev_u_offset + IDX(i,3,k)]
                            )/59.0/48.0*64*dev_dy[0];
    output[IDX(i,4,k)] =  (     dev_var_in[*dev_u_offset + IDX(i,7,k)]
                                -  6.0*dev_var_in[*dev_u_offset + IDX(i,6,k)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(i,5,k)]
                                - 10.0*dev_var_in[*dev_u_offset + IDX(i,4,k)]
                                +  3.0*dev_var_in[*dev_u_offset + IDX(i,3,k)]
                                )/43.0/48.0*64*dev_dy[0];
    output[IDX(i,5,k)] =  (     dev_var_in[*dev_u_offset + IDX(i,8,k)]
                                -  6.0*dev_var_in[*dev_u_offset + IDX(i,7,k)]
                                + 15.0*dev_var_in[*dev_u_offset + IDX(i,6,k)]
                                - 19.0*dev_var_in[*dev_u_offset + IDX(i,5,k)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(i,4,k)]
                                -  3.0*dev_var_in[*dev_u_offset + IDX(i,3,k)]
                                )/49.0/48.0*64*dev_dy[0];
    }

   if ((*dev_bflag & (1u<<OCT_DIR_UP)) && (j == 4)) {
       
       const int je = ny - 3;
       output[IDX(i,je-3,k)] = (dev_var_in[*dev_u_offset + IDX(i,je-6,k)]
                                -  6.0*dev_var_in[*dev_u_offset + IDX(i,je-5,k)]
                                + 15.0*dev_var_in[*dev_u_offset + IDX(i,je-4,k)]
                                - 19.0*dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                                -  3.0*dev_var_in[*dev_u_offset + IDX(i,je-1,k)]
                                )/49.0/48.0*64*dev_dy[0];
        
        output[IDX(i,je-2,k)] = (dev_var_in[*dev_u_offset + IDX(i,je-5,k)]
                                -  6.0*dev_var_in[*dev_u_offset + IDX(i,je-4,k)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                                - 10.0*dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                                +  3.0*dev_var_in[*dev_u_offset + IDX(i,je-1,k)]
                                )/43.0/48.0*64*dev_dy[0];
       
 
        output[IDX(i,je-1,k)] = ( dev_var_in[*dev_u_offset + IDX(i,je-4,k)]
                                -  3.0*dev_var_in[*dev_u_offset + IDX(i,je-3,k)]
                                +  3.0*dev_var_in[*dev_u_offset + IDX(i,je-2,k)]
                                -      dev_var_in[*dev_u_offset + IDX(i,je-1,k)]
                                )/59.0/48.0*64*dev_dy[0];
   }
}

void cuda_ko_deriv42_y(double * output, double * dev_var_in, 
   int * dev_u_offset, double * dev_dy, int * dev_sz,
   int* dev_bflag, const unsigned int * host_sz)
{
   hipError_t cudaStatus;
   const int ie = host_sz[0] - 3;//x direction
   const int je = host_sz[1] - 3;//y direction
   const int ke = host_sz[2] - 3;//z direction

   int temp_max = (ie>je)? ie : je;
   int maximumIterations = (temp_max>ke) ? temp_max: ke;

   int requiredBlocks = (9+maximumIterations) / 10;
  
   calc_ko_deriv42_y <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                    dim3((ie + requiredBlocks -1)/requiredBlocks,
                    (je + requiredBlocks -1)/requiredBlocks, 
                    (ke + requiredBlocks -1)/requiredBlocks) >>> (output, dev_var_in,
                        dev_dy, dev_bflag, dev_sz, dev_u_offset);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_ko_deriv42_y kernal!\n", cudaStatus);
            return;
    }

}

__global__ void calc_ko_deriv42_z(double * output, double * dev_var_in,
    double *dev_dz, int* dev_bflag, int* dev_sz, int* dev_u_offset) {
   
   //ib, jb, kb values are accumulated to the x, y, z
   int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
   int j = 3 + threadIdx.y + blockIdx.y * blockDim.y;
   int k = 3 + threadIdx.z + blockIdx.z * blockDim.z;

   int nx = dev_sz[0];
   int ny = dev_sz[1];

   if(i >= nx-3 || j >= ny-3 || k >= dev_sz[2]-3) return;

   int pp = IDX(i, j, k);
   int n = nx * ny;
   
   output[pp] = (-1.0 / 64.0 / dev_dz[0]) *
                (
                    -      dev_var_in[*dev_u_offset + pp-3*n]
                    +  6.0*dev_var_in[*dev_u_offset + pp-2*n]
                    - 15.0*dev_var_in[*dev_u_offset + pp-n]
                    + 20.0*dev_var_in[*dev_u_offset + pp]
                    - 15.0*dev_var_in[*dev_u_offset + pp+n]
                    +  6.0*dev_var_in[*dev_u_offset + pp+2*n]
                    -      dev_var_in[*dev_u_offset + pp+3*n]
                    );

   
   if ((*dev_bflag & (1u<<OCT_DIR_DOWN)) && (k == 3)) {

    output[IDX(i,3,k)] =  (      dev_var_in[*dev_u_offset +IDX(i,k,6)]
                                - 3.0*dev_var_in[*dev_u_offset +IDX(i,k,5)]
                                + 3.0*dev_var_in[*dev_u_offset + IDX(i,k,4)]
                                -     dev_var_in[*dev_u_offset + IDX(i,k,3)]
                            )/59.0/48.0*64*dev_dz[0];
    output[IDX(i,j,4)] =  (     dev_var_in[*dev_u_offset + IDX(i,j,7)]
                                -  6.0*dev_var_in[*dev_u_offset + IDX(i,j,6)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(i,j,5)]
                                - 10.0*dev_var_in[*dev_u_offset + IDX(i,j,4)]
                                +  3.0*dev_var_in[*dev_u_offset + IDX(i,j,3)]
                                )/43.0/48.0*64*dev_dz[0];
    output[IDX(i,j,5)] =  (     dev_var_in[*dev_u_offset + IDX(i,j,8)]
                                -  6.0*dev_var_in[*dev_u_offset + IDX(i,j,7)]
                                + 15.0*dev_var_in[*dev_u_offset + IDX(i,j,6)]
                                - 19.0*dev_var_in[*dev_u_offset + IDX(i,j,5)]
                                + 12.0*dev_var_in[*dev_u_offset + IDX(i,j,4)]
                                -  3.0*dev_var_in[*dev_u_offset + IDX(i,j,3)]
                                )/49.0/48.0*64*dev_dz[0];
    }

   if ((*dev_bflag & (1u<<OCT_DIR_UP)) && (k == 4)) {
       
       const int ke = dev_sz[2] - 3;
       output[IDX(i,j,ke-3)] = (    dev_var_in[*dev_u_offset + IDX(i,j,ke-6)]
                                    -  6.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-5)]
                                    + 15.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-4)]
                                    - 19.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                                    + 12.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                                    -  3.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-1)]
                                    )/49.0/48.0*64*dev_dz[0];
        
        output[IDX(i,j,ke-2)] = (   dev_var_in[*dev_u_offset + IDX(i,j,ke-5)]
                                    -  6.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-4)]
                                    + 12.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                                    - 10.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                                    +  3.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-1)]
                                    )/43.0/48.0*64*dev_dz[0];
       
 
        output[IDX(i,j,ke-1)] = (   dev_var_in[*dev_u_offset + IDX(i,j,ke-4)]
                                    -  3.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-3)]
                                    +  3.0*dev_var_in[*dev_u_offset + IDX(i,j,ke-2)]
                                    -      dev_var_in[*dev_u_offset + IDX(i,j,ke-1)]
                                    )/59.0/48.0*64*dev_dz[0];
   }
}

void cuda_ko_deriv42_z(double * output, double * dev_var_in, 
   int * dev_u_offset, double * dev_dz, int * dev_sz,
   int* dev_bflag, const unsigned int * host_sz)
{
   hipError_t cudaStatus;
   const int ie = host_sz[0] - 3;//x direction
   const int je = host_sz[1] - 3;//y direction
   const int ke = host_sz[2] - 3;//z direction

   int temp_max = (ie>je)? ie : je;
   int maximumIterations = (temp_max>ke) ? temp_max: ke;
   
   int requiredBlocks = (9+maximumIterations) / 10;
  
   calc_ko_deriv42_z <<< dim3(requiredBlocks, requiredBlocks, requiredBlocks),
                    dim3((ie + requiredBlocks -1)/requiredBlocks,
                    (je + requiredBlocks -1)/requiredBlocks, 
                    (ke + requiredBlocks -1)/requiredBlocks) >>> (output, dev_var_in,
                        dev_dz, dev_bflag, dev_sz, dev_u_offset);
   
   cudaStatus = hipDeviceSynchronize();
   if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cuda_ko_deriv42_z kernal!\n", cudaStatus);
        return;
   }
}