#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernal_deriv42_x.h"
#include <math.h>
#include "def.h"


__inline __device__ int IDX_R(int i,int j,int k,int *nx,int *ny) 
{
    return (i) + (*nx) * ( (j) + (*ny) * (k) );
}

__global__ void calc_deriv42_x(double *dev_Dxu, double *dev_u,
                                double* dev_idx_by_2, double* dev_idx_by_12,
                                 int* dev_ie, int* dev_je, int* dev_ke, int* dev_flag, int* dev_nx, int* dev_ny ) {

    //ib, jb, kb values are accumulated to the x, y, z
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 1 + threadIdx.y + blockIdx.x * blockDim.y;
    int k = 1 + threadIdx.z + blockIdx.x * blockDim.z;

    if(i <= *dev_ie && j <= *dev_je && k <= *dev_ke) {
        
        int pp = IDX_R(i, j, k, dev_nx, dev_ny);
        dev_Dxu[pp] = (dev_u[pp - 2] - 8.0 * dev_u[pp - 1] + 
                        8.0 * dev_u[pp + 1] - dev_u[pp + 2]) * *dev_idx_by_12;
        
        if (*dev_flag == 0 && (i == 3 || j == 4)) {
            dev_Dxu[IDX_R(3, j, k, dev_nx, dev_ny)] = (-3.0 * dev_u[IDX_R(3, j, k, dev_nx, dev_ny)]
                                        + 4.0 * dev_u[IDX_R(4, j, k, dev_nx, dev_ny)]
                                        - dev_u[IDX_R(5, j, k, dev_nx, dev_ny)]
                                        ) * *dev_idx_by_2;
            dev_Dxu[IDX_R(4, j, k, dev_nx, dev_ny)] = (-dev_u[IDX_R(3, j, k, dev_nx, dev_ny)]
                                        + dev_u[IDX_R(5, j, k, dev_nx, dev_ny)]
                                        ) * *dev_idx_by_2;
        }

        if (*dev_flag == 1 && (i == *dev_ie - 2 || i == *dev_ie - 1)) {
            dev_Dxu[IDX_R(*dev_ie - 2, j, k, dev_nx, dev_ny)] = (-dev_u[IDX_R(*dev_ie - 3, j, k, dev_nx, dev_ny)]
                                            + dev_u[IDX_R(*dev_ie - 1, j, k, dev_nx, dev_ny)]
                                            ) * *dev_idx_by_2;

            dev_Dxu[IDX_R(*dev_ie - 1, j, k, dev_nx, dev_ny)] = (dev_u[IDX_R(*dev_ie - 3, j, k, dev_nx, dev_ny)]
                                            - 4.0 * dev_u[IDX_R(*dev_ie - 2, j, k, dev_nx, dev_ny)]
                                            + 3.0 * dev_u[IDX_R(*dev_ie - 1, j, k, dev_nx, dev_ny)]
                                            ) * *dev_idx_by_2;
        }
    }
    
}



void kernal_calc_deriv42_x(double *const Dxu, const double *const u,
                           const double dx, const unsigned int *sz, unsigned bflag) {

    const double idx = 1.0 / dx;
    const double idx_by_2 = 0.5 * idx;
    const double idx_by_12 = idx / 12.0;

    const int nx = sz[0];
    const int ny = sz[1];
    const int nz = sz[2];
    // const int ib = 3;
    // const int jb = 1;
    // const int kb = 1;
    const int ie = nx - 3;
    const int je = ny - 1;
    const int ke = nz - 1;
    int flag = 10;
    int *dev_flag, *dev_ie, *dev_je, *dev_ke, *dev_nx, *dev_ny;
    double *dev_idx, *dev_idx_by_2, *dev_idx_by_12;
    double *dev_Dxu, *dev_u;

    hipMalloc((void**) &dev_idx, sizeof(double));
    hipMalloc((void**) &dev_idx_by_2, sizeof(double));
    hipMalloc((void**) &dev_idx_by_12, sizeof(double));
    hipMalloc((void**) &dev_ie, sizeof(int));
    hipMalloc((void**) &dev_je, sizeof(int));
    hipMalloc((void**) &dev_ke, sizeof(int));
    hipMalloc((void**) &dev_u, sizeof(double)*sizeof(u));
    hipMalloc((void**) &dev_Dxu, sizeof(double)*sizeof(Dxu));
    hipMalloc((void**) &dev_nx, sizeof(int));
    hipMalloc((void**) &dev_ny, sizeof(int));

    if (bflag & (1u<<OCT_DIR_DOWN)) {
        flag = 0;
    }
    
    if (bflag & (1u<<OCT_DIR_UP)) {
        flag = 1;
    }

    hipMemcpy(dev_idx, &idx, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_idx_by_2, &idx_by_2, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_idx_by_12, &idx_by_12, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_flag, &flag, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_ie, &ie, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_je, &je, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_ke, &ke, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_Dxu, Dxu, sizeof(double) * sizeof(Dxu), hipMemcpyHostToDevice);
    hipMemcpy(dev_u, u, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_nx, &nx, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_ny, &ny, sizeof(int), hipMemcpyHostToDevice);

    int maximumIterations = max(max(ie, je), ke);
    int requiredBlocks = maximumIterations / 10;
    if (ie % 10 == 0 || je % 10 == 0 || ke % 10 == 0) {
        requiredBlocks++;
    }
    
    int threads_x = ie / requiredBlocks;
    int threads_y = je / requiredBlocks;
    int threads_z = ke / requiredBlocks;

    calc_deriv42_x <<< 1000, dim3(threads_x,threads_y,threads_z) >>> (dev_Dxu, dev_u, dev_idx_by_2,
                         dev_idx_by_12, dev_ie, dev_je, dev_ke, dev_flag, dev_nx, dev_ny);
                    
    hipMemcpy(Dxu, dev_Dxu, sizeof(double)*sizeof(Dxu), hipMemcpyDeviceToHost);

    
}