#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernal_deriv42_x.h"
#include <math.h>
#include <iostream>
using namespace std;

#define IDX(i,j,k) ( )

__device__ int IDX(int i,int j,int k,int *nx,int *ny){
    return (i) + nx * ( (j) + ny * (k) ) ;
}

__global__ void calc_deriv42_x(double *dev_Dxu, double *dev_u,
                                double* dev_idx_by_2, double* dev_idx_by_12,
                                 int* dev_ie, int* dev_je, int* dev_ke, int* dev_flag ) {

    //ib, jb, kb values are accumulated to the x, y, z
    int i = 3 + threadIdx.x + blockIdx.x * blockDim.x;
    int j = 1 + threadIdx.y + blockIdx.x * blockDim.y;
    int k = 1 + threadIdx.z + blockIdx.x * blockDim.z;

    if(i <= *dev_ie && j <= *dev_je && k <= *dev_ke) {
        int pp = IDX(i, j, k);
        dev_Dxu[pp] = (dev_u[pp - 2] - 8.0 * dev_u[pp - 1] + 
                        8.0 * dev_u[pp + 1] - dev_u[pp + 2]) * dev_idx_by_12;
        
        if (*dev_flag == 0 && (i == 3 || j == 4)) {
            dev_Dxu[IDX(3, j, k)] = (-3.0 * dev_u[IDX(3, j, k)]
                                        + 4.0 * dev_u[IDX(4, j, k)]
                                        - dev_u[IDX(5, j, k)]
                                        ) * *dev_idx_by_2;
            dev_Dxu[IDX(4, j, k)] = (-dev_u[IDX(3, j, k)]
                                        + dev_u[IDX(5, j, k)]
                                        ) * *dev_idx_by_2;
        }

        if (*dev_flag == 1 && (i == *dev_ie - 2 || i == *dev_ie - 1)) {
            dev_Dxu[IDX(dev_ie - 2, j, k)] = (-dev_u[IDX(*dev_ie - 3, j, k)]
                                            + dev_u[IDX(*dev_ie - 1, j, k)]
                                            ) * *dev_idx_by_2;

            dev_Dxu[IDX(*dev_ie - 1, j, k)] = (dev_u[IDX(*dev_ie - 3, j, k)]
                                            - 4.0 * dev_u[IDX(*dev_ie - 2, j, k)]
                                            + 3.0 * dev_u[IDX(*dev_ie - 1, j, k)]
                                            ) * *dev_idx_by_2;
        }
    }
    
}



void kernal_calc_deriv42_x(double *const Dxu, const double *const u,
                           const double dx, const unsigned int *sz, unsigned bflag) {

    const double idx = 1.0 / dx;
    const double idx_by_2 = 0.5 * idx;
    const double idx_by_12 = idx / 12.0;

    const int nx = sz[0];
    const int ny = sz[1];
    const int nz = sz[2];
    const int ib = 3;
    const int jb = 1;
    const int kb = 1;
    const int ie = nx - 3;
    const int je = ny - 1;
    const int ke = nz - 1;
    int flag = -1;
    int *dev_flag, *dev_ie, *dev_je, *dev_ke;
    double *dev_idx, *dev_idx_by_2, *dev_idx_by_12;
    double *dev_Dxu, *dev_u;

    hipMalloc((void**) &dev_idx, sizeof(double));
    hipMalloc((void**) &dev_idx_by_2, sizeof(double));
    hipMalloc((void**) &dev_idx_by_12, sizeof(double));
    hipMalloc((void**) &dev_ie, sizeof(int));
    hipMalloc((void**) &dev_je, sizeof(int));
    hipMalloc((void**) &dev_ke, sizeof(int));
    hipMalloc((void**) &dev_u, sizeof(double)*sizeof(u));
    hipMalloc((void**) &dev_Dxu, sizeof(double)*sizeof(Dxu));


    if (bflag & (1u<<OCT_DIR_DOWN)) {
        flag = 0;
    }
    
    if (bflag & (1u<<OCT_DIR_UP)) {
        flag = 1;
    }

    hipMemcpy(dev_idx, &idx, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_idx_by_2, &idx_by_2, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_idx_by_12, &idx_by_12, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_flag, &flag, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_ie, &ie, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_je, &je, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_ke, &ke, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_Dxu, Dxu, sizeof(double) * sizeof(Dxu), hipMemcpyHostToDevice);
    hipMemcpy(dev_u, u, sizeof(int), hipMemcpyHostToDevice);

    //threads per block is set to 500
    int requiredNumberOfThreads = ie * je * ke;
    // int numberOfBlocks = ceil(requiredNumberOfThreads / 500.0);
    // int threads_x = ceil(ie/(numberOfBlocks*1.0));
    // int threads_y = ceil(je/(numberOfBlocks*1.0));
    // int threads_z = ceil(ke/(numberOfBlocks*1.0));
    int threadsPerBlock_x = ie / 10;
    int threadsPerBlock_y = je / 10;
    int threadsPerBlock_z = ke / 10;

    int requiredNumberOfBlocks = max(max(threadsPerBlock_x,threadsPerBlock_y),threadsPerBlock_z);

    if (requiredNumberOfThreads % 1000 != 0) {
        requiredNumberOfBlocks++;
    }

    calc_deriv42_x <<< 1000, dim3(10,10,10) >>> (dev_Dxu, dev_u, dev_idx_by_2,
                         dev_idx_by_12, dev_ie, dev_je, dev_ke, dev_flag);
                    
    hipMemcpy(Dxu,dev_Dxu,sizeof(double)*sizeof(Dxu),hipMemcpyDeviceToHost);

    
}